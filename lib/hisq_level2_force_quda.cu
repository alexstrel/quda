#include "hip/hip_runtime.h"
#include <read_gauge.h>
#include <gauge_quda.h>

#include "hisq_force_quda.h"
#include "force_common.h"
#include "hw_quda.h"
#include "hisq_force_macros.h"


namespace hisq {
  namespace fermion_force {


#define LOAD_ANTI_HERMITIAN LOAD_ANTI_HERMITIAN_SINGLE



#define LOAD_MATRIX(src, dir, idx, var) LOAD_MATRIX_12_SINGLE(src, dir, idx, var)

#define FF_SITE_MATRIX_LOAD_TEX 1

#if (FF_SITE_MATRIX_LOAD_TEX == 1)
#define linkEvenTex siteLink0TexSingle_recon
#define linkOddTex siteLink1TexSingle_recon
#define FF_LOAD_MATRIX(src, dir, idx, var) LOAD_MATRIX_12_SINGLE_TEX(src##Tex, dir, idx, var)
#else
#define FF_LOAD_MATRIX(src, dir, idx, var) LOAD_MATRIX_12_SINGLE(src, dir, idx, var)
#endif


#define LOAD_ARRAY_12_SINGLE_TEX(gauge, dir, idx, var)do{                     \
      var[0] = tex1Dfetch(gauge, idx + dir*Vhx3);                         \
      var[1] = tex1Dfetch(gauge, idx + dir*Vhx3 + Vh);                    \
      var[2] = tex1Dfetch(gauge, idx + dir*Vhx3 + Vhx2);                  \
    }while(0)

#define FF_LOAD_ARRAY(src, dir, idx, var) LOAD_ARRAY_12_SINGLE_TEX(src##Tex, dir, idx, var)    


//void load_array_12_from_texture(float4 array[4], int dir, int idx) 

// Need to compute the neighbouring index
// Can really clean the code up
// I need to force the compiler to inline these functions

__forceinline__ __device__ 
void computeNewFullIndexPlusUpdate(int dir, int idx, int new_x[4], int & new_index){
  switch(dir){
    case 0:
      new_index = ((new_x[0]==X1m1)?idx-X1m1:idx+1);		
      new_x[0] = (new_x[0]==X1m1)?0:new_x[0]+1;                       
    break;

    case 1:                                                        
      new_index = ( (new_x[1]==X2m1)?idx-X2X1mX1:idx+X1);		
      new_x[1] = (new_x[1]==X2m1)?0:new_x[1]+1;    
    break;     

    case 2:                                                         
      new_index = ( (new_x[2]==X3m1)?idx-X3X2X1mX2X1:idx+X2X1);	
      new_x[2] = (new_x[2]==X3m1)?0:new_x[2]+1;                         
    break;                              

    case 3:                                                         
      new_index = ( (new_x[3]==X4m1)?idx-X4X3X2X1mX3X2X1:idx+X3X2X1); 
      new_x[3] = (new_x[3]==X4m1)?0:new_x[3]+1;                        
    break;      

    case 4:    
      new_index = ( (new_x[0]==0)?idx+X1m1:idx-1);                 
      new_x[0] = (new_x[0]==0)?X1m1:new_x[0] - 1;                       
    break;                                                      

    case 5:                                                         
      new_index = ( (new_x[1]==0)?idx+X2X1mX1:idx-X1);              
      new_x[1] = (new_x[1]==0)?X2m1:new_x[1] - 1;                       
    break;                                                      
  }
  return;
}


#define FF_COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(mydir, idx, new_idx) do {	\
  switch(mydir){                                                  \
    case 0:                                                         \
                                                                    new_idx = ( (new_x[0]==X1m1)?idx-X1m1:idx+1);			\
    new_x[0] = (new_x[0]==X1m1)?0:new_x[0]+1;                         \
    break;                                                      \
    case 1:                                                         \
                                                                    new_idx = ( (new_x[1]==X2m1)?idx-X2X1mX1:idx+X1);		\
    new_x[1] = (new_x[1]==X2m1)?0:new_x[1]+1;                         \
    break;                                                      \
    case 2:                                                         \
                                                                    new_idx = ( (new_x[2]==X3m1)?idx-X3X2X1mX2X1:idx+X2X1);	\
    new_x[2] = (new_x[2]==X3m1)?0:new_x[2]+1;                         \
    break;                                                      \
    case 3:                                                         \
                                                                    new_idx = ( (new_x[3]==X4m1)?idx-X4X3X2X1mX3X2X1:idx+X3X2X1); \
    new_x[3] = (new_x[3]==X4m1)?0:new_x[3]+1;                         \
    break;                                                      \
  }                                                               \
}while(0)

#define FF_COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(mydir, idx, new_idx) do {	\
  switch(mydir){                                                  \
    case 0:                                                         \
                                                                    new_idx = ( (new_x[0]==0)?idx+X1m1:idx-1);			\
    new_x[0] = (new_x[0]==0)?X1m1:new_x[0] - 1;                       \
    break;                                                      \
    case 1:                                                         \
                                                                    new_idx = ( (new_x[1]==0)?idx+X2X1mX1:idx-X1);		\
    new_x[1] = (new_x[1]==0)?X2m1:new_x[1] - 1;                       \
    break;                                                      \
    case 2:                                                         \
                                                                    new_idx = ( (new_x[2]==0)?idx+X3X2X1mX2X1:idx-X2X1);		\
    new_x[2] = (new_x[2]==0)?X3m1:new_x[2] - 1;                       \
    break;                                                      \
    case 3:                                                         \
                                                                    new_idx = ( (new_x[3]==0)?idx+X4X3X2X1mX3X2X1:idx-X3X2X1);	\
    new_x[3] = (new_x[3]==0)?X4m1:new_x[3] - 1;                       \
    break;                                                      \
  }                                                               \
}while(0)



#define FF_COMPUTE_NEW_FULL_IDX_PLUS(old_x1, old_x2, old_x3, old_x4, idx, mydir, new_idx) do { \
  switch(mydir){                                                  \
    case 0:                                                         \
                                                                    new_idx = ( (old_x1==X1m1)?idx-X1m1:idx+1);			\
    break;                                                      \
    case 1:                                                         \
                                                                    new_idx = ( (old_x2==X2m1)?idx-X2X1mX1:idx+X1);		\
    break;                                                      \
    case 2:                                                         \
                                                                    new_idx = ( (old_x3==X3m1)?idx-X3X2X1mX2X1:idx+X2X1);	\
    break;                                                      \
    case 3:                                                         \
                                                                    new_idx = ( (old_x4==X4m1)?idx-X4X3X2X1mX3X2X1:idx+X3X2X1); \
    break;                                                      \
  }                                                               \
}while(0)

#define FF_COMPUTE_NEW_FULL_IDX_MINUS(old_x1, old_x2, old_x3, old_x4, idx, mydir, new_idx) do { \
  switch(mydir){                                                  \
    case 0:                                                         \
                                                                    new_idx = ( (old_x1==0)?idx+X1m1:idx-1);			\
    break;                                                      \
    case 1:                                                         \
                                                                    new_idx = ( (old_x2==0)?idx+X2X1mX1:idx-X1);		\
    break;                                                      \
    case 2:                                                         \
                                                                    new_idx = ( (old_x3==0)?idx+X3X2X1mX2X1:idx-X2X1);		\
    break;                                                      \
    case 3:                                                         \
                                                                    new_idx = ( (old_x4==0)?idx+X4X3X2X1mX3X2X1:idx-X3X2X1);	\
    break;                                                      \
  }                                                               \
}while(0)





//this macro require link_W, link_X and ah variables defined
#define SIMPLE_MAT_FORCE_TO_MOM(mat, mom, idx, dir, temp_mat) do { \
  {                                                             \
  float2 AH0, AH1, AH2, AH3, AH4;                               \
  LOAD_ANTI_HERMITIAN(mom, dir, idx, AH);			\
  UNCOMPRESS_ANTI_HERMITIAN(ah, temp_mat);			\
  SCALAR_MULT_ADD_SU3_MATRIX(temp_mat, mat, 1.0, link_W);	\
  MAKE_ANTI_HERMITIAN(temp_mat, ah);				\
  WRITE_ANTI_HERMITIAN_SINGLE(mom, dir, idx, AH);		\
  }                                                             \
}while(0)






// Struct to determine the coefficient sign at compile time
template<int pos_dir, int odd_lattice>
struct CoeffSign
{
    static const int result = -1;
};

template<>
struct CoeffSign<0,0>
{
    static const int result = 1;
};

template<>
struct CoeffSign<1,1>
{
    static const int result = 1;
};




__device__ void reconstructSign(int* const sign, int dir, int i[4]){
  *sign=1;
  switch(dir){
    case XUP:
      if( (i[3]&1)==1) *sign=1;
    break;

    case YUP:
      if( ((i[3]+i[0])&1) == 1) *sign=1; 
    break;

    case ZUP:
      if( ((i[3]+i[0]+i[1])&1) == 1) *sign=1; 
    break;

    case TUP:
      if(i[3] == X4m1) *sign=1; 
    break;
  }
}



void
hisq_force_init_cuda(QudaGaugeParam* param)
{
  static int fermion_force_init_cuda_flag = 0; 

  if (fermion_force_init_cuda_flag){
    return;
  }
  fermion_force_init_cuda_flag=1;
  init_kernel_cuda(param);    
}




template<int oddBit>
  __global__ void 
do_compute_force_kernel(float4* linkEven, float4* linkOdd,
    float2* momMatrixEven, float2* momMatrixOdd,
    int sig,
    float2* momEven, float2* momOdd)
{
  int sid = blockIdx.x * blockDim.x + threadIdx.x;

  int x[4];
  int z1 = sid/X1h;
  int x1h = sid - z1*X1h;
  int z2 = z1/X2;
  x[1] = z1 - z2*X2;
  x[3] = z2/X3;
  x[2] = z2 - x[3]*X3;
  int x1odd = (x[1] + x[2] + x[3] + oddBit) & 1;
  x[0] = 2*x1h + x1odd;
  int X = 2*sid + x1odd;

  int link_sign;

  float4 LINK_W[5];
  float2 COLOR_MAT_W[9];
  float2 COLOR_MAT_X[9];

  FF_LOAD_ARRAY(linkEven, sig, sid, LINK_W);
  reconstructSign(&link_sign, sig, x);	
  RECONSTRUCT_LINK_12(sig, sid, link_sign, link_W);

  LOAD_MOM_MATRIX_SINGLE(momMatrixEven, sig, sid, COLOR_MAT_X);
  MAT_MUL_MAT(link_W, color_mat_X, color_mat_W);

  SIMPLE_MAT_FORCE_TO_MOM(color_mat_W, momEven, sid, sig, link_W);
}





template<int sig_positive, int mu_positive, int oddBit> 
__global__ void
do_middle_link_kernel(const float2 * const tempEven, 
    float2 * const PmuOdd, float2 * const P3Even,
    const float2 * const QprevOdd, 		
    float2 * const QmuEven, 
    int sig, int mu, float coeff,
    float4 * const linkEven, float4 * const linkOdd,
    float2 * const momEven,
    float2 * const momMatrixEven 
    ) 
{		
  int sid = blockIdx.x * blockDim.x + threadIdx.x;

  int x[4];
  int z1 = sid/X1h;
  int x1h = sid - z1*X1h;
  int z2 = z1/X2;
  x[1] = z1 - z2*X2;
  x[3] = z2/X3;
  x[2] = z2 - x[3]*X3;
  int x1odd = (x[1] + x[2] + x[3] + oddBit) & 1;
  x[0] = 2*x1h + x1odd;
  int X = 2*sid + x1odd;

  int new_x[4];
  int new_mem_idx;
  int ad_link_sign=1;
  int ab_link_sign=1;
  int bc_link_sign=1;

  float4 LINK_W[5];
  float4 LINK_X[5];
  float4 LINK_Y[5];
  float4 LINK_Z[5];


  float2 COLOR_MAT_W[9];
  float2 COLOR_MAT_Y[9];
  float2 COLOR_MAT_X[9];
  float2 COLOR_MAT_Z[9];

//  float2 AH0, AH1, AH2, AH3, AH4;

  //        A________B
  //    mu   |      |
  // 	    D|      |C
  //	  
  //	  A is the current point (sid)
  int point_b, point_c, point_d;
  int ad_link_nbr_idx, ab_link_nbr_idx, bc_link_nbr_idx;
  int mymu;

  new_x[0] = x[0];
  new_x[1] = x[1];
  new_x[2] = x[2];
  new_x[3] = x[3];

  if(mu_positive){
    mymu =mu;
    FF_COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(mu, X, new_mem_idx);
  }else{
    mymu = OPP_DIR(mu);
   // computeNewFullIndexPlusUpdate(OPP_DIR(mu),X,new_x,new_mem_idx);
    FF_COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(OPP_DIR(mu), X, new_mem_idx);	
  }
  point_d = (new_mem_idx >> 1);
  if (mu_positive){
    ad_link_nbr_idx = point_d;
    reconstructSign(&ad_link_sign, mymu, new_x);
  }else{
    ad_link_nbr_idx = sid;
    reconstructSign(&ad_link_sign, mymu, x);	
  }


  int mysig; 
  if(sig_positive){
    mysig = sig;
    FF_COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(sig, new_mem_idx, new_mem_idx);
  }else{
    mysig = OPP_DIR(sig);
    FF_COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(OPP_DIR(sig), new_mem_idx, new_mem_idx);	
  }
  point_c = (new_mem_idx >> 1);
  if (mu_positive){
    bc_link_nbr_idx = point_c;	
    reconstructSign(&bc_link_sign, mymu, new_x);
  }
  // So far, we have just computed ad_link_nbr_idx and 
  // bc_link_nbr_idx

  new_x[0] = x[0];
  new_x[1] = x[1];
  new_x[2] = x[2];
  new_x[3] = x[3];
  if(sig_positive){
    FF_COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(sig, X, new_mem_idx);
  }else{
    FF_COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(OPP_DIR(sig), X, new_mem_idx);	
  }
  point_b = (new_mem_idx >> 1); 

  if (!mu_positive){
    bc_link_nbr_idx = point_b;
    reconstructSign(&bc_link_sign, mymu, new_x);
  }   

  if(sig_positive){
    ab_link_nbr_idx = sid;
    reconstructSign(&ab_link_sign, mysig, x);	
  }else{	
    ab_link_nbr_idx = point_b;
    reconstructSign(&ab_link_sign, mysig, new_x);
  }
  // now we have ab_link_nbr_idx


  // load the link variable connecting a and b 
  // Store in link_W 
  if(sig_positive){
    FF_LOAD_ARRAY(linkEven, mysig, ab_link_nbr_idx, LINK_W);	
  }else{
    FF_LOAD_ARRAY(linkOdd, mysig, ab_link_nbr_idx, LINK_W);	
  }
  RECONSTRUCT_LINK_12(mysig, ab_link_nbr_idx, ab_link_sign, link_W);

  // load the link variable connecting b and c 
  // Store in link_X
  if(mu_positive){
    FF_LOAD_ARRAY(linkEven, mymu, bc_link_nbr_idx, LINK_X);
  }else{ 
    FF_LOAD_ARRAY(linkOdd, mymu, bc_link_nbr_idx, LINK_X);	
  }
  RECONSTRUCT_LINK_12(mymu, bc_link_nbr_idx, bc_link_sign, link_X);



  LOAD_MATRIX_18_SINGLE(tempEven, point_c, COLOR_MAT_Y);
  // I do not think that Q3 is needed!
  if(mu_positive){
    ADJ_MAT_MUL_MAT(link_X, color_mat_Y, color_mat_W);
  }else{
    MAT_MUL_MAT(link_X, color_mat_Y, color_mat_W);
  }


  // Why write to PmuOdd instead of PmuEven?
  // Well, PmuEven would require tempOdd 
  // i.e., an extra device-memory access
  WRITE_MATRIX_18_SINGLE(PmuOdd, point_b, COLOR_MAT_W);
  if(sig_positive){
    MAT_MUL_MAT(link_W, color_mat_W, color_mat_Y);
  }else{ 
    ADJ_MAT_MUL_MAT(link_W, color_mat_W, color_mat_Y);
  }
  WRITE_MATRIX_18_SINGLE(P3Even, sid, COLOR_MAT_Y);


  if(mu_positive){
    FF_LOAD_ARRAY(linkOdd, mymu, ad_link_nbr_idx, LINK_Y);
    RECONSTRUCT_LINK_12(mymu, ad_link_nbr_idx, ad_link_sign, link_Y);
  }else{
    FF_LOAD_ARRAY(linkEven, mymu, ad_link_nbr_idx, LINK_X);
    RECONSTRUCT_LINK_12(mymu, ad_link_nbr_idx, ad_link_sign, link_X);
    ADJ_MAT(link_X, link_Y);
  }


  // if threeStaple - additional factorisation here!
  if(QprevOdd == NULL){
    if(sig_positive){
      MAT_MUL_MAT(color_mat_W, link_Y, color_mat_Y);
    }
    ASSIGN_MAT(link_Y, color_mat_W); 
    WRITE_MATRIX_18_SINGLE(QmuEven, sid, COLOR_MAT_W);
  }else{ 
    LOAD_MATRIX_18_SINGLE(QprevOdd, point_d, COLOR_MAT_Y);   
    MAT_MUL_MAT(color_mat_Y, link_Y, color_mat_X);
    WRITE_MATRIX_18_SINGLE(QmuEven, sid, COLOR_MAT_X);
    if(sig_positive){
      MAT_MUL_MAT(color_mat_W, color_mat_X, color_mat_Y);
    }	
  }

   
  if(sig_positive){
   const float & mycoeff = -CoeffSign<sig_positive,oddBit>::result*coeff;

   LOAD_MOM_MATRIX_SINGLE(momMatrixEven, sig, sid, COLOR_MAT_Z);
   SCALAR_MULT_ADD_SU3_MATRIX(color_mat_Z, color_mat_Y, mycoeff, color_mat_Z);
   WRITE_MOM_MATRIX_SINGLE(momMatrixEven, sig, sid, COLOR_MAT_Z);
  }

  return;
}



  static void 
compute_force_kernel(float4* linkEven, float4* linkOdd, FullGauge cudaSiteLink,
    float2* momMatrixEven, float2* momMatrixOdd,
    int sig, dim3 gridDim, dim3 blockDim,
    float2* momEven, float2* momOdd)
{
  dim3 halfGridDim(gridDim.x/2, 1, 1);

  // Need to see if this is necessary in the lates version of quda
  hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);
  hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.odd,  cudaSiteLink.bytes);

  do_compute_force_kernel<0><<<halfGridDim, blockDim>>>(linkEven, linkOdd,
      momMatrixEven, momMatrixOdd,
      sig, 
      momEven, momOdd);
  hipUnbindTexture(siteLink0TexSingle_recon);
  hipUnbindTexture(siteLink1TexSingle_recon);

  hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
  hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);

  do_compute_force_kernel<1><<<halfGridDim, blockDim>>>(linkOdd, linkEven,
      momMatrixOdd, momMatrixEven,
      sig,
      momOdd, momEven);

  hipUnbindTexture(siteLink0TexSingle_recon);
  hipUnbindTexture(siteLink1TexSingle_recon);

}





  static void
middle_link_kernel(const float2 * const tempEven, const float2 * const tempOdd, 
    float2 * const PmuEven,   float2 * const PmuOdd,
    float2 * const P3Even,    float2 * const P3Odd,
    const float2 * const QprevEven, const float2 * const QprevOdd,
    float2 * const QmuEven,   float2 * const QmuOdd,
    int sig, int mu, float coeff,
    float4 * const linkEven, float4 * const linkOdd, FullGauge cudaSiteLink,
    float2 * const  momEven, float2 * const momOdd,
    dim3 gridDim, dim3 BlockDim,
    float2 * const momMatrixEven, float2 * const momMatrixOdd)
{
  dim3 halfGridDim(gridDim.x/2, 1,1);

  hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);
  hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);

  if (GOES_FORWARDS(sig) && GOES_FORWARDS(mu)){	
    do_middle_link_kernel<1,1,0><<<halfGridDim, BlockDim>>>( tempEven,
        PmuOdd,  P3Even,
        QprevOdd,
        QmuEven, 
        sig, mu, coeff,
        linkEven, linkOdd,
        momEven, 
        momMatrixEven);
    hipUnbindTexture(siteLink0TexSingle_recon);
    hipUnbindTexture(siteLink1TexSingle_recon);
    //opposite binding
    hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
    hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);

    do_middle_link_kernel<1,1,1><<<halfGridDim, BlockDim>>>( tempOdd, 
        PmuEven,  P3Odd,
        QprevEven,
        QmuOdd, 
        sig, mu, coeff,
        linkOdd, linkEven,
        momOdd, 
        momMatrixOdd);
  }else if (GOES_FORWARDS(sig) && GOES_BACKWARDS(mu)){
    do_middle_link_kernel<1,0,0><<<halfGridDim, BlockDim>>>( tempEven,
        PmuOdd,  P3Even,
        QprevOdd,
        QmuEven,
        sig, mu, coeff,
        linkEven, linkOdd,
        momEven, 
        momMatrixEven);	
    hipUnbindTexture(siteLink0TexSingle_recon);
    hipUnbindTexture(siteLink1TexSingle_recon);

    //opposite binding
    hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
    hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);

    do_middle_link_kernel<1,0,1><<<halfGridDim, BlockDim>>>( tempOdd, 
        PmuEven,  P3Odd,
        QprevEven,
        QmuOdd,  
        sig, mu, coeff,
        linkOdd, linkEven,
        momOdd, 
        momMatrixOdd);

  }else if (GOES_BACKWARDS(sig) && GOES_FORWARDS(mu)){
    do_middle_link_kernel<0,1,0><<<halfGridDim, BlockDim>>>( tempEven, 
        PmuOdd,  P3Even,
        QprevOdd,
        QmuEven, 
        sig, mu, coeff,
        linkEven, linkOdd,
        momEven, 
        momMatrixEven);	
    hipUnbindTexture(siteLink0TexSingle_recon);
    hipUnbindTexture(siteLink1TexSingle_recon);

    //opposite binding
    hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
    hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);

    do_middle_link_kernel<0,1,1><<<halfGridDim, BlockDim>>>( tempOdd,
        PmuEven,  P3Odd,
        QprevEven, 
        QmuOdd, 
        sig, mu, coeff,
        linkOdd, linkEven,
        momOdd, 
        momMatrixOdd);
  }else{
    do_middle_link_kernel<0,0,0><<<halfGridDim, BlockDim>>>( tempEven,
        PmuOdd, P3Even,
        QprevOdd,
        QmuEven, 
        sig, mu, coeff,
        linkEven, linkOdd,
        momEven, 
        momMatrixEven);		

    hipUnbindTexture(siteLink0TexSingle_recon);
    hipUnbindTexture(siteLink1TexSingle_recon);

    //opposite binding
    hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
    hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);

    do_middle_link_kernel<0,0,1><<<halfGridDim, BlockDim>>>( tempOdd, 
        PmuEven,  P3Odd,
        QprevEven,
        QmuOdd,  
        sig, mu, coeff,
        linkOdd, linkEven,
        momOdd, 
        momMatrixOdd);		
  }
  hipUnbindTexture(siteLink0TexSingle_recon);
  hipUnbindTexture(siteLink1TexSingle_recon);    
}


template<int sig_positive, int mu_positive, int oddBit>
  __global__ void
do_side_link_kernel(const float2 * const P3Even, 
    const float2* const TempxEven, const float2 * const TempxOdd,
    float2 * const shortPOdd,
    int sig, int mu, float coeff, float accumu_coeff,
    const float4 * const linkEven, const float4 * const linkOdd,
    float2 * const momEven, float2 * const momOdd,
    float2 * const momMatrixEven, float2 * const momMatrixOdd)
{

  int sid = blockIdx.x * blockDim.x + threadIdx.x;

  int x[4];
  int z1 = sid/X1h;
  int x1h = sid - z1*X1h;
  int z2 = z1/X2;
  x[1] = z1 - z2*X2;
  x[3] = z2/X3;
  x[2] = z2 - x[3]*X3;
  int x1odd = (x[1] + x[2] + x[3] + oddBit) & 1;
  x[0] = 2*x1h + x1odd;
  int X = 2*sid + x1odd;

  int ad_link_sign = 1;

  float4 LINK_W[5];
  float2 COLOR_MAT_W[9], COLOR_MAT_X[9], COLOR_MAT_Y[9], COLOR_MAT_Z[9];

 
  /*    
   * 	  compute the side link contribution to the momentum
   *

   sig
   A________B
   |      |   mu
   D |      |C

   A is the current point (sid)
   */

  float mycoeff;
  int point_d;
  int ad_link_nbr_idx;
  int mymu;
  int new_mem_idx;

  int new_x[4];
  new_x[0] = x[0];
  new_x[1] = x[1];
  new_x[2] = x[2];
  new_x[3] = x[3];

  if(mu_positive){
    mymu=mu;
    FF_COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(mymu,X, new_mem_idx);
  }else{
    mymu = OPP_DIR(mu);
    FF_COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(mymu, X, new_mem_idx);
  }
  point_d = (new_mem_idx >> 1);


  if (mu_positive){
    ad_link_nbr_idx = point_d;
    reconstructSign(&ad_link_sign, mymu, new_x);
  }else{
    ad_link_nbr_idx = sid;
    reconstructSign(&ad_link_sign, mymu, x);	
  }


  LOAD_MATRIX_18_SINGLE(P3Even, sid, COLOR_MAT_Y);
  if(mu_positive){
    FF_LOAD_ARRAY(linkOdd, mymu, ad_link_nbr_idx, LINK_W);
  }else{
    FF_LOAD_ARRAY(linkEven, mymu, ad_link_nbr_idx, LINK_W);
  }

  RECONSTRUCT_LINK_12(mymu, ad_link_nbr_idx, ad_link_sign, link_W);	


  // Should all be inside if (shortPOdd)
  if (shortPOdd){
    if (mu_positive){
      MAT_MUL_MAT(link_W, color_mat_Y, color_mat_W);
    }else{
      ADJ_MAT_MUL_MAT(link_W, color_mat_Y, color_mat_W);
    }
    LOAD_MATRIX_18_SINGLE(shortPOdd, point_d, COLOR_MAT_X);
    SCALAR_MULT_ADD_MATRIX(color_mat_X, color_mat_W, accumu_coeff, color_mat_X);
    WRITE_MATRIX_18_SINGLE(shortPOdd, point_d, COLOR_MAT_X);
  }


  mycoeff = CoeffSign<sig_positive,oddBit>::result*coeff;

  if (mu_positive){
    if(TempxOdd){
      LOAD_MATRIX_18_SINGLE(TempxOdd, point_d, COLOR_MAT_X);
      MAT_MUL_MAT(color_mat_Y, color_mat_X, color_mat_W);
    }else{
      ASSIGN_MAT(color_mat_Y, color_mat_W);
    }
   
    LOAD_MOM_MATRIX_SINGLE(momMatrixOdd, mu, point_d, COLOR_MAT_Z);
    SCALAR_MULT_ADD_SU3_MATRIX(color_mat_Z, color_mat_W, mycoeff, color_mat_Z);
    WRITE_MOM_MATRIX_SINGLE(momMatrixOdd, mu, point_d, COLOR_MAT_Z);
  }else{

    if(TempxOdd){
      LOAD_MATRIX_18_SINGLE(TempxOdd, point_d, COLOR_MAT_X);
      ADJ_MAT(color_mat_X,color_mat_W);
      MAT_MUL_ADJ_MAT(color_mat_W, color_mat_Y, color_mat_X);
    }else{
      ADJ_MAT(color_mat_Y, color_mat_X);
    }
    
    LOAD_MOM_MATRIX_SINGLE(momMatrixEven, OPP_DIR(mu), sid, COLOR_MAT_Z);
    SCALAR_MULT_ADD_SU3_MATRIX(color_mat_Z, color_mat_X, mycoeff, color_mat_Z);
    WRITE_MOM_MATRIX_SINGLE(momMatrixEven, OPP_DIR(mu), sid, COLOR_MAT_Z);
  }
  return;
}





static void
side_link_kernel(float2* P3Even, float2* P3Odd, 
		 float2* TempxEven, float2* TempxOdd,
		 float2* shortPEven,  float2* shortPOdd,
		 int sig, int mu, float coeff, float accumu_coeff,
		 float4* linkEven, float4* linkOdd, FullGauge cudaSiteLink,
		 float2* momEven, float2* momOdd,
		 dim3 gridDim, dim3 blockDim,
		 float2* momMatrixEven, float2* momMatrixOdd)
{
    dim3 halfGridDim(gridDim.x/2,1,1);
    
    hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);
    hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);   

    if (GOES_FORWARDS(sig) && GOES_FORWARDS(mu)){
	do_side_link_kernel<1,1,0><<<halfGridDim, blockDim>>>( P3Even, 
							       TempxEven,  TempxOdd,
							       shortPOdd,
							       sig, mu, coeff, accumu_coeff,
							       linkEven, linkOdd,
							       momEven, momOdd,
							       momMatrixEven, momMatrixOdd);
	hipUnbindTexture(siteLink0TexSingle_recon);
	hipUnbindTexture(siteLink1TexSingle_recon);

	//opposite binding
	hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
	hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);

	do_side_link_kernel<1,1,1><<<halfGridDim, blockDim>>>( P3Odd, 
							       TempxOdd,  TempxEven,
							       shortPEven,
							       sig, mu, coeff, accumu_coeff,
							       linkOdd, linkEven,
							       momOdd, momEven,
							       momMatrixOdd, momMatrixEven);
	
    }else if (GOES_FORWARDS(sig) && GOES_BACKWARDS(mu)){
	do_side_link_kernel<1,0,0><<<halfGridDim, blockDim>>>( P3Even, 
							       TempxEven,  TempxOdd,
							       shortPOdd,
							       sig, mu, coeff, accumu_coeff,
							       linkEven,  linkOdd,
							       momEven, momOdd,
							       momMatrixEven, momMatrixOdd);		
	hipUnbindTexture(siteLink0TexSingle_recon);
	hipUnbindTexture(siteLink1TexSingle_recon);

	//opposite binding
	hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
	hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);

	do_side_link_kernel<1,0,1><<<halfGridDim, blockDim>>>( P3Odd, 
							       TempxOdd,  TempxEven,
							       shortPEven,
							       sig, mu, coeff, accumu_coeff,
							       linkOdd, linkEven,
							       momOdd, momEven,
							       momMatrixOdd, momMatrixEven);		

    }else if (GOES_BACKWARDS(sig) && GOES_FORWARDS(mu)){
	do_side_link_kernel<0,1,0><<<halfGridDim, blockDim>>>( P3Even,
							       TempxEven,  TempxOdd,
							       shortPOdd,
							       sig, mu, coeff, accumu_coeff,
							       linkEven,  linkOdd,
							       momEven, momOdd,
							       momMatrixEven, momMatrixOdd);
	hipUnbindTexture(siteLink0TexSingle_recon);
	hipUnbindTexture(siteLink1TexSingle_recon);

	//opposite binding
	hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
	hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);

	do_side_link_kernel<0,1,1><<<halfGridDim, blockDim>>>( P3Odd,
							       TempxOdd,  TempxEven,
							       shortPEven,
							       sig, mu, coeff, accumu_coeff,
							       linkOdd, linkEven,
							       momOdd, momEven,
							       momMatrixOdd, momMatrixEven);
	
    }else{
	do_side_link_kernel<0,0,0><<<halfGridDim, blockDim>>>( P3Even,
							       TempxEven,  TempxOdd,
							       shortPOdd,
							       sig, mu, coeff, accumu_coeff,
							       linkEven, linkOdd,
							       momEven, momOdd,
							       momMatrixEven, momMatrixOdd);
	hipUnbindTexture(siteLink0TexSingle_recon);
	hipUnbindTexture(siteLink1TexSingle_recon);

	//opposite binding
	hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
	hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);
	
	do_side_link_kernel<0,0,1><<<halfGridDim, blockDim>>>( P3Odd, 
							       TempxOdd,  TempxEven,
							       shortPEven,
							       sig, mu, coeff, accumu_coeff,
							       linkOdd, linkEven,
							       momOdd, momEven,
							       momMatrixOdd, momMatrixEven);
    }
    
    hipUnbindTexture(siteLink0TexSingle_recon);
    hipUnbindTexture(siteLink1TexSingle_recon);    

}

template<int sig_positive, int mu_positive, int oddBit>
__global__ void
do_all_link_kernel(const float2* tempEven, 
		float2* QprevOdd,
		float2* PmuEven, float2* PmuOdd,
		float2* P3Even, float2* P3Odd,
		float2* P3muEven, float2* P3muOdd,
		float2* shortPEven, float2* shortPOdd,
		int sig, int mu, 
		float coeff, float mcoeff, float accumu_coeff,
		float4* linkEven, float4* linkOdd,
		float2* momEven, float2* momOdd,
		float2* momMatrixEven, float2* momMatrixOdd)
{
    int sid = blockIdx.x * blockDim.x + threadIdx.x;

    int x[4];

    int z1 = sid/X1h;
    int x1h = sid - z1*X1h;
    int z2 = z1/X2;
    x[1] = z1 - z2*X2;
    x[3] = z2/X3;
    x[2] = z2 - x[3]*X3;
    int x1odd = (x[1] + x[2] + x[3] + oddBit) & 1;
    x[0] = 2*x1h + x1odd;
    int X = 2*sid + x1odd;
    
    int new_x[4];
    int ad_link_sign=1;
    int ab_link_sign=1;
    int bc_link_sign=1;   
    
    float4 LINK_W[5], LINK_X[5], LINK_Y[5], LINK_Z[5];
    float2 COLOR_MAT_W[9], COLOR_MAT_Y[9], COLOR_MAT_X[9], COLOR_MAT_Z[9];
 

    /*       sig
           A________B
	mu  |      |
	  D |      |C
	  
	  A is the current point (sid)
    */
    int point_b, point_c, point_d;
    int ad_link_nbr_idx, ab_link_nbr_idx, bc_link_nbr_idx;
    int mymu;
    int new_mem_idx;
    new_x[0] = x[0];
    new_x[1] = x[1];
    new_x[2] = x[2];
    new_x[3] = x[3];

    if(mu_positive){
	mymu =mu;
	FF_COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(mu, X, new_mem_idx);
    }else{
	mymu = OPP_DIR(mu);
	FF_COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(OPP_DIR(mu), X, new_mem_idx);	
    }
    point_d = (new_mem_idx >> 1);

    if (mu_positive){
	ad_link_nbr_idx = point_d;
	reconstructSign(&ad_link_sign, mymu, new_x);
    }else{
	ad_link_nbr_idx = sid;
	reconstructSign(&ad_link_sign, mymu, x);	
    }
  
 
    int mysig; 
    if(sig_positive){
	mysig = sig;
	FF_COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(sig, new_mem_idx, new_mem_idx);
    }else{
	mysig = OPP_DIR(sig);
	FF_COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(OPP_DIR(sig), new_mem_idx, new_mem_idx);	
    }
    point_c = (new_mem_idx >> 1);
    if (mu_positive){
	bc_link_nbr_idx = point_c;	
	reconstructSign(&bc_link_sign, mymu, new_x);
    }
    
    new_x[0] = x[0];
    new_x[1] = x[1];
    new_x[2] = x[2];
    new_x[3] = x[3];
    if(sig_positive){
	FF_COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(sig, X, new_mem_idx);
    }else{
	FF_COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(OPP_DIR(sig), X, new_mem_idx);	
    }
    point_b = (new_mem_idx >> 1);
    if (!mu_positive){
	bc_link_nbr_idx = point_b;
	reconstructSign(&bc_link_sign, mymu, new_x);
    }      
    
    if(sig_positive){
	ab_link_nbr_idx = sid;
	reconstructSign(&ab_link_sign, mysig, x);	
    }else{	
	ab_link_nbr_idx = point_b;
	reconstructSign(&ab_link_sign, mysig, new_x);
    }

    LOAD_MATRIX_18_SINGLE(QprevOdd, point_d, COLOR_MAT_X);
    ASSIGN_MAT(color_mat_X, link_W);
  
    if (mu_positive){
	FF_LOAD_ARRAY(linkOdd, mymu, ad_link_nbr_idx, LINK_Y);
    }else{
	FF_LOAD_ARRAY(linkEven, mymu, ad_link_nbr_idx, LINK_Y);
    }
    RECONSTRUCT_LINK_12(mymu, ad_link_nbr_idx, ad_link_sign, link_Y);

    if (mu_positive){
	MAT_MUL_MAT(link_W, link_Y, color_mat_W);
    }else{
	MAT_MUL_ADJ_MAT(link_W, link_Y, color_mat_W);
    }
    LOAD_MATRIX_18_SINGLE(tempEven, point_c, COLOR_MAT_Y);


    if (mu_positive){
	FF_LOAD_ARRAY(linkEven, mymu, bc_link_nbr_idx, LINK_W);
    }else{
	FF_LOAD_ARRAY(linkOdd, mymu, bc_link_nbr_idx, LINK_W);	
    }
    RECONSTRUCT_LINK_12(mymu, bc_link_nbr_idx, bc_link_sign, link_W);


    if (mu_positive){    
	ADJ_MAT_MUL_MAT(link_W, color_mat_Y, link_X);
    }else{
	MAT_MUL_MAT(link_W, color_mat_Y, link_X);
    }
    // link_X now connects site b to the outer product! 
    // Done with LINK_W for the time being.	

    if (sig_positive){
	FF_LOAD_ARRAY(linkEven, mysig, ab_link_nbr_idx, LINK_W);
    }else{
	FF_LOAD_ARRAY(linkOdd, mysig, ab_link_nbr_idx, LINK_W);
    }
    RECONSTRUCT_LINK_12(mysig, ab_link_nbr_idx, ab_link_sign, link_W);


   if (sig_positive){        
     MAT_MUL_MAT(link_W, link_X, color_mat_Y);
   }else{
     ADJ_MAT_MUL_MAT(link_W, link_X, color_mat_Y);
   }
    // color_mat_Y now connects site a to the outer product 
    // Force from the forward link in the staple


   const float & mycoeff = CoeffSign<sig_positive,oddBit>::result*coeff;
   if (sig_positive)
   {	
     MAT_MUL_MAT(link_X, color_mat_W, link_Z);
     ASSIGN_MAT(link_Z, color_mat_W);
     LOAD_MOM_MATRIX_SINGLE(momMatrixEven, sig, sid, COLOR_MAT_Z);
     SCALAR_MULT_ADD_SU3_MATRIX(color_mat_Z, link_Z, mycoeff, color_mat_Z);
     WRITE_MOM_MATRIX_SINGLE(momMatrixEven, sig, sid, COLOR_MAT_Z);
   }

   // Note that mu is known at compile time 
   // Surely this code can be made more generic
   // QprevOdd = color_mat_X
   if (mu_positive)
   {
     MAT_MUL_MAT(color_mat_Y, color_mat_X, link_Z);
     LOAD_MOM_MATRIX_SINGLE(momMatrixOdd, mu, point_d, COLOR_MAT_Z);
     SCALAR_MULT_ADD_SU3_MATRIX(color_mat_Z, link_Z, mycoeff, color_mat_Z);
     WRITE_MOM_MATRIX_SINGLE(momMatrixOdd, mu, point_d, COLOR_MAT_Z);
     MAT_MUL_MAT(link_Y, color_mat_Y, color_mat_W);	
   }else
   {
     ADJ_MAT_MUL_ADJ_MAT(color_mat_X, color_mat_Y, link_Z);	
     LOAD_MOM_MATRIX_SINGLE(momMatrixEven, OPP_DIR(mu), sid, COLOR_MAT_Z);
     SCALAR_MULT_ADD_SU3_MATRIX(color_mat_Z, link_Z, mycoeff, color_mat_Z);
     WRITE_MOM_MATRIX_SINGLE(momMatrixEven, OPP_DIR(mu), sid, COLOR_MAT_Z);
     ADJ_MAT_MUL_MAT(link_Y, color_mat_Y, color_mat_W);	
   }

   LOAD_MATRIX_18_SINGLE(shortPOdd, point_d, COLOR_MAT_Y);
   SCALAR_MULT_ADD_MATRIX(color_mat_Y, color_mat_W, accumu_coeff, color_mat_Y);
   WRITE_MATRIX_18_SINGLE(shortPOdd, point_d, COLOR_MAT_Y);

   return;
}



static void
all_link_kernel(const float2* link_ZxEven, const float2* link_ZxOdd,
		float2* QprevEven, float2* QprevOdd, 
		float2* PmuEven, float2* PmuOdd,
		float2* P3Even, float2* P3Odd,
		float2* P3muEven, float2* P3muOdd,
		float2* shortPEven, float2* shortPOdd,
		int sig, int mu,
		float coeff, float mcoeff, float accumu_coeff,
		float4* linkEven, float4* linkOdd, FullGauge cudaSiteLink,
		float2* momEven, float2* momOdd,
		dim3 gridDim, dim3 blockDim,
		float2* momMatrixEven, float2* momMatrixOdd)
		   
{
    dim3 halfGridDim(gridDim.x/2, 1,1);

    hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);
    hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
    
    if (GOES_FORWARDS(sig) && GOES_FORWARDS(mu)){		
	do_all_link_kernel<1,1,0><<<halfGridDim, blockDim>>>( link_ZxEven,  
							      QprevOdd, 
							      PmuEven,  PmuOdd,
							      P3Even,  P3Odd,
							      P3muEven,  P3muOdd,
							      shortPEven,  shortPOdd,
							      sig,  mu,
							      coeff, mcoeff, accumu_coeff,
							      linkEven, linkOdd,
							      momEven, momOdd,
							      momMatrixEven, momMatrixOdd);
	hipUnbindTexture(siteLink0TexSingle_recon);
	hipUnbindTexture(siteLink1TexSingle_recon);

	//opposite binding
	hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
	hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);
	do_all_link_kernel<1,1,1><<<halfGridDim, blockDim>>>( link_ZxOdd,  
							      QprevEven,
							      PmuOdd,  PmuEven,
							      P3Odd,  P3Even,
							      P3muOdd,  P3muEven,
							      shortPOdd,  shortPEven,
							      sig,  mu,
							      coeff, mcoeff, accumu_coeff,
							      linkOdd, linkEven,
							      momOdd, momEven,
							      momMatrixOdd, momMatrixEven);	

	
    }else if (GOES_FORWARDS(sig) && GOES_BACKWARDS(mu)){

	do_all_link_kernel<1,0,0><<<halfGridDim, blockDim>>>( link_ZxEven,   
							      QprevOdd,
							      PmuEven,  PmuOdd,
							      P3Even,  P3Odd,
							      P3muEven,  P3muOdd,
							      shortPEven,  shortPOdd,
							      sig,  mu, 
							      coeff, mcoeff, accumu_coeff,
							      linkEven, linkOdd,
							      momEven, momOdd,
							      momMatrixEven, momMatrixOdd);	
	hipUnbindTexture(siteLink0TexSingle_recon);
	hipUnbindTexture(siteLink1TexSingle_recon);

	//opposite binding
	hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
	hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);

	do_all_link_kernel<1,0,1><<<halfGridDim, blockDim>>>( link_ZxOdd,  
							      QprevEven, 
							      PmuOdd,  PmuEven,
							      P3Odd,  P3Even,
							      P3muOdd,  P3muEven,
							      shortPOdd,  shortPEven,
							      sig,  mu, 
							      coeff, mcoeff, accumu_coeff,
							      linkOdd, linkEven,
							      momOdd, momEven,
							      momMatrixOdd, momMatrixEven);	
	
    }else if (GOES_BACKWARDS(sig) && GOES_FORWARDS(mu)){
	do_all_link_kernel<0,1,0><<<halfGridDim, blockDim>>>( link_ZxEven,  
							      QprevOdd, 
							      PmuEven,  PmuOdd,
							      P3Even,  P3Odd,
							      P3muEven,  P3muOdd,
							      shortPEven,  shortPOdd,
							      sig,  mu, 
							      coeff, mcoeff, accumu_coeff,
							      linkEven, linkOdd,
							      momEven, momOdd, 
							      momMatrixEven, momMatrixOdd);	
	hipUnbindTexture(siteLink0TexSingle_recon);
	hipUnbindTexture(siteLink1TexSingle_recon);

	//opposite binding
	hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
	hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);

	
	do_all_link_kernel<0,1,1><<<halfGridDim, blockDim>>>( link_ZxOdd,  
							      QprevEven, 
							      PmuOdd,  PmuEven,
							      P3Odd,  P3Even,
							      P3muOdd,  P3muEven,
							      shortPOdd,  shortPEven,
							      sig,  mu, 
							      coeff, mcoeff, accumu_coeff,
							      linkOdd, linkEven,
							      momOdd, momEven,
							      momMatrixOdd, momMatrixEven);		
    }else{
	do_all_link_kernel<0,0,0><<<halfGridDim, blockDim>>>( link_ZxEven, 
							      QprevOdd, 
							      PmuEven,  PmuOdd,
							      P3Even,  P3Odd,
							      P3muEven,  P3muOdd,
							      shortPEven,  shortPOdd,
							      sig,  mu, 
							      coeff, mcoeff, accumu_coeff,
							      linkEven, linkOdd,
							      momEven, momOdd,
							      momMatrixEven, momMatrixOdd);	

	hipUnbindTexture(siteLink0TexSingle_recon);
	hipUnbindTexture(siteLink1TexSingle_recon);

	//opposite binding
	hipBindTexture(0, siteLink0TexSingle_recon, cudaSiteLink.odd, cudaSiteLink.bytes);
	hipBindTexture(0, siteLink1TexSingle_recon, cudaSiteLink.even, cudaSiteLink.bytes);

	do_all_link_kernel<0,0,1><<<halfGridDim, blockDim>>>( link_ZxOdd,  
							      QprevEven, 
							      PmuOdd,  PmuEven,
							      P3Odd,  P3Even,
							      P3muOdd,  P3muEven,
							      shortPOdd,  shortPEven,
							      sig,  mu, 
							      coeff, mcoeff, accumu_coeff,
							      linkOdd, linkEven,
							      momOdd, momEven,
							      momMatrixOdd, momMatrixEven);	
    }

    hipUnbindTexture(siteLink0TexSingle_recon);
    hipUnbindTexture(siteLink1TexSingle_recon);
}

/*
__global__ void
one_and_naik_terms_kernel(float2* TempxEven, float2* TempxOdd,
			  float2* PmuEven,   float2* PmuOdd, 
			  float2* PnumuEven, float2* PnumuOdd,
			  int mu, float OneLink, float Naik, float mNaik,
			  float4* linkEven, float4* linkOdd,
			  float2* momEven, float2* momOdd)
{
    int sid = blockIdx.x * blockDim.x + threadIdx.x;
    int oddBit = 0;
    float2* myTempx = TempxEven;
    float2* myPmu = PmuEven;
    float2* myPnumu = PnumuEven;
    float2* myMom = momEven;
    float4* myLink = linkEven;    
    float2* otherTempx = TempxOdd;
    float2* otherPnumu = PnumuOdd;
    float4* otherLink = linkOdd;
    
    float2 HWA0, HWA1, HWA2, HWA3, HWA4, HWA5;
    float2 HWB0, HWB1, HWB2, HWB3, HWB4, HWB5;
    float2 HWC0, HWC1, HWC2, HWC3, HWC4, HWC5;
    float2 HWD0, HWD1, HWD2, HWD3, HWD4, HWD5;
    float4 LINK_W0, LINK_W1, LINK_W2, LINK_W3, LINK_W4;
    float4 LINK_X0, LINK_X1, LINK_X2, LINK_X3, LINK_X4;
    float2 AH0, AH1, AH2, AH3, AH4;    
    
    if (sid >= Vh){
        oddBit =1;
        sid -= Vh;
	
	myTempx = TempxOdd;
	myPmu = PmuOdd;
	myPnumu = PnumuOdd;
	myMom = momOdd;
	myLink = linkOdd;  	
	otherTempx = TempxEven;
	otherPnumu = PnumuEven;
	otherLink = linkEven;
    }
    
    int z1 = sid/X1h;
    int x1h = sid - z1*X1h;
    int z2 = z1/X2;
    int x2 = z1 - z2*X2;
    int x4 = z2/X3;
    int x3 = z2 - x4*X3;
    int x1odd = (x2 + x3 + x4 + oddBit) & 1;
    int x1 = 2*x1h + x1odd;
    //int X = 2*sid + x1odd;
    
    int dx[4];
    int new_x[0], new_x[1], new_x[2], new_x[3], new_idx;
    int sign=1;
    
    if (GOES_BACKWARDS(mu)){
	//The one link
	LOAD_HW(myPmu, sid, HWA);
	LOAD_HW(myTempx, sid, HWB);
	ADD_FORCE_TO_MOM(hwa, hwb, myMom, sid, OPP_DIR(mu), OneLink, oddBit);
	
	//Naik term
	dx[3]=dx[2]=dx[1]=dx[0]=0;
	dx[OPP_DIR(mu)] = -1;
	new_x[0] = (x1 + dx[0] + X1)%X1;
	new_x[1] = (x2 + dx[1] + X2)%X2;
	new_x[2] = (x3 + dx[2] + X3)%X3;
	new_x[3] = (x4 + dx[3] + X4)%X4;	
	new_idx = (new_x[3]*X3X2X1+new_x[2]*X2X1+new_x[1]*X1+new_x[0]) >> 1;
	LOAD_HW(otherTempx, new_idx, HWA);
	LOAD_MATRIX(otherLink, OPP_DIR(mu), new_idx, LINK_W);
	reconstructSign(sign, OPP_DIR(mu), new_x[0],new_x[1],new_x[2],new_x[3]);
	RECONSTRUCT_LINK_12(OPP_DIR(mu), new_idx, sign, link_W);		
	ADJ_MAT_MUL_HW(link_W, hwa, hwc); //Popmu
	
	LOAD_HW(myPnumu, sid, HWD);
	ADD_FORCE_TO_MOM(hwd, hwc, myMom, sid, OPP_DIR(mu), mNaik, oddBit);
	
	dx[3]=dx[2]=dx[1]=dx[0]=0;
	dx[OPP_DIR(mu)] = 1;
	new_x[0] = (x1 + dx[0] + X1)%X1;
	new_x[1] = (x2 + dx[1] + X2)%X2;
	new_x[2] = (x3 + dx[2] + X3)%X3;
	new_x[3] = (x4 + dx[3] + X4)%X4;	
	new_idx = (new_x[3]*X3X2X1+new_x[2]*X2X1+new_x[1]*X1+new_x[0]) >> 1;
	LOAD_HW(otherPnumu, new_idx, HWA);
	LOAD_MATRIX(myLink, OPP_DIR(mu), sid, LINK_W);
	reconstructSign(sign, OPP_DIR(mu), x1, x2, x3, x4);
	RECONSTRUCT_LINK_12(OPP_DIR(mu), sid, sign, link_W);	
	MAT_MUL_HW(link_W, hwa, hwc);
	ADD_FORCE_TO_MOM(hwc, hwb, myMom, sid, OPP_DIR(mu), Naik, oddBit);	
    }else{
	dx[3]=dx[2]=dx[1]=dx[0]=0;
	dx[mu] = 1;
	new_x[0] = (x1 + dx[0] + X1)%X1;
	new_x[1] = (x2 + dx[1] + X2)%X2;
	new_x[2] = (x3 + dx[2] + X3)%X3;
	new_x[3] = (x4 + dx[3] + X4)%X4;	
	new_idx = (new_x[3]*X3X2X1+new_x[2]*X2X1+new_x[1]*X1+new_x[0]) >> 1;
	LOAD_HW(otherTempx, new_idx, HWA);
	LOAD_MATRIX(myLink, mu, sid, LINK_W);
	reconstructSign(sign, mu, x1, x2, x3, x4);
	RECONSTRUCT_LINK_12(mu, sid, sign, link_W);
	MAT_MUL_HW(link_W, hwa, hwb);
	
	LOAD_HW(myPnumu, sid, HWC);
	ADD_FORCE_TO_MOM(hwb, hwc, myMom, sid, mu, Naik, oddBit);
	

    }
}
*/


#define Pmu 	  tempmat[0]
#define P3        tempmat[1]
#define P5	  tempmat[2]
#define Pnumu     tempmat[3]
#define P3mu	  tempmat[3]
#define P5nu	  tempmat[3]
#define P7 	  tempmat[3]
#define Prhonumu  tempmat[3]
#define P7rho     tempmat[3]


// Here, we have a problem. 
// If we use float2 to store the Ps and float2 
// for the Qs. We can't use the same link_Zoraries 
// here. 
// I wonder which is better? 
// To use float2 for both Ps and Qs and 
// and use the same link_Zorary matrices 
// or use float2 for the Ps and float4 for the 
// Qs and use separate sets of matrices?
// Ultimately, I will use float4 for the Q matrices 
// for the first level of smearing and float2 
// for the Q matrices for the second level of smearing. 
// To begin with, use float4 for everything.
// Note, I will have to go back and undo the float4s above.


// if first level of smearing
 #define Qmu      tempCmat[0]
 #define Qnumu    tempCmat[1]
 #define Qrhonumu tempCmat[2] 
 #define Q5       tempCmat[2]


// tempCmat should be a full compressed matrix
// FullCompMat

// if !first level of smearing
//#define Qmu	  tempmat[7]
//#define Qnumu	  tempmat[8]
//#define Qrhonumu  tempmat[2] // same as Prhonumu

// Need to define new types 
// FullMat 
// FullCompMat

template<typename Real>
static void
do_hisq_force_cuda(Real eps, Real weight1, Real weight2,  Real* act_path_coeff, FullOprod cudaOprod, // need to change this code
		      FullGauge cudaSiteLink, FullMom cudaMom, FullGauge cudaMomMatrix, FullMatrix tempmat[7], FullMatrix tempCmat[4], QudaGaugeParam* param)
{
    
    int mu, nu, rho, sig;
    float coeff;
    
    float OneLink, Lepage, Naik, FiveSt, ThreeSt, SevenSt;
    float mLepage, mNaik, mFiveSt, mThreeSt, mSevenSt;
    
    Real ferm_epsilon;
    ferm_epsilon = 2.0*weight1*eps;
    OneLink = act_path_coeff[0]*ferm_epsilon ;
    Naik    = act_path_coeff[1]*ferm_epsilon ; mNaik    = -Naik;
    ThreeSt = act_path_coeff[2]*ferm_epsilon ; mThreeSt = -ThreeSt;
    FiveSt  = act_path_coeff[3]*ferm_epsilon ; mFiveSt  = -FiveSt;
    SevenSt = act_path_coeff[4]*ferm_epsilon ; mSevenSt = -SevenSt;
    Lepage  = act_path_coeff[5]*ferm_epsilon ; mLepage  = -Lepage;
    
    int DirectLinks[8] ;    
    
    for(mu=0;mu<8;mu++){
        DirectLinks[mu] = 0 ;
    }
        
    int volume = param->X[0]*param->X[1]*param->X[2]*param->X[3];
    dim3 blockDim(BLOCK_DIM,1,1);
    dim3 gridDim(volume/blockDim.x, 1, 1);
   
    int null = -1;
   
    for(sig=0; sig < 8; sig++){
        for(mu = 0; mu < 8; mu++){
            if ( (mu == sig) || (mu == OPP_DIR(sig))){
                continue;
            }
	    //3-link
	    //Kernel A: middle link
	   
	    middle_link_kernel( (float2*)cudaOprod.even.data[OPP_DIR(sig)], (float2*)cudaOprod.odd.data[OPP_DIR(sig)],
				(float2*)Pmu.even.data, (float2*)Pmu.odd.data,
				(float2*)P3.even.data, (float2*)P3.odd.data,
				(float2*)NULL,         (float2*)NULL,
				(float2*)Qmu.even.data, (float2*)Qmu.odd.data,
				sig, mu, mThreeSt,
				(float4*)cudaSiteLink.even, (float4*)cudaSiteLink.odd, cudaSiteLink, 
				(float2*)cudaMom.even, (float2*)cudaMom.odd, 
				gridDim, blockDim,
			        (float2*)cudaMomMatrix.even, (float2*)cudaMomMatrix.odd); // I have added true and false to indicate 
							                                  // whether I am on a three staple
                                                                                          // Actually, I just have to check if the pointer   
                                                                                          // to the previous path is a NULL pointer!     



	
	    checkCudaError();

            for(nu=0; nu < 8; nu++){
                if (nu == sig || nu == OPP_DIR(sig)
                    || nu == mu || nu == OPP_DIR(mu)){
                    continue;
                }

		//5-link: middle link
		//Kernel B
		middle_link_kernel( (float2*)Pmu.even.data, (float2*)Pmu.odd.data,
				    (float2*)Pnumu.even.data, (float2*)Pnumu.odd.data,
				    (float2*)P5.even.data, (float2*)P5.odd.data,
				    (float2*)Qmu.even.data, (float2*)Qmu.odd.data, // input Q matrix
				    (float2*)Qnumu.even.data, (float2*)Qnumu.odd.data,
				    sig, nu, FiveSt,
				    (float4*)cudaSiteLink.even, (float4*)cudaSiteLink.odd, cudaSiteLink, 
				    (float2*)cudaMom.even, (float2*)cudaMom.odd,
				    gridDim, blockDim,
				    (float2*)cudaMomMatrix.even, (float2*)cudaMomMatrix.odd); // no longer on a threeStaple => have to read in Qprev

		checkCudaError();

                for(rho =0; rho < 8; rho++){
                    if (rho == sig || rho == OPP_DIR(sig)
                        || rho == mu || rho == OPP_DIR(mu)
                        || rho == nu || rho == OPP_DIR(nu)){
                        continue;
                    }
		    //7-link: middle link and side link
		    //kernel C
		    if(FiveSt != 0)coeff = SevenSt/FiveSt ; else coeff = 0;
		    all_link_kernel((float2*)Pnumu.even.data, (float2*)Pnumu.odd.data,
				    (float2*)Qnumu.even.data, (float2*)Qnumu.odd.data,
				    (float2*)Prhonumu.even.data, (float2*)Prhonumu.odd.data,
				    (float2*)P7.even.data, (float2*)P7.odd.data,
				    (float2*)P7rho.even.data, (float2*)P7rho.odd.data,
				    (float2*)P5.even.data, (float2*)P5.odd.data,
				    sig, rho, SevenSt, mSevenSt, coeff,
				    (float4*)cudaSiteLink.even, (float4*)cudaSiteLink.odd, cudaSiteLink,
				    (float2*)cudaMom.even, (float2*)cudaMom.odd,
				    gridDim, blockDim,
				    (float2*)cudaMomMatrix.even, (float2*)cudaMomMatrix.odd);	
		    checkCudaError();

		}//rho  		
                // P7, P7rho, P7rhonumu are free to be used again


		//5-link: side link
		//kernel B2
		if(ThreeSt != 0)coeff = FiveSt/ThreeSt; else coeff = 0;
		side_link_kernel((float2*)P5.even.data, (float2*)P5.odd.data,
		//		 (float2*)P5nu.even.data, (float2*)P5nu.odd.data, // output
				 (float2*)Qmu.even.data, (float2*)Qmu.odd.data,
	         //	        (float2*)Qnumu.even.data, (float2*)Qnumu.odd.data,
				 (float2*)P3.even.data, (float2*)P3.odd.data,
				 sig, nu, mFiveSt, coeff,
				 (float4*)cudaSiteLink.even, (float4*)cudaSiteLink.odd, cudaSiteLink,
				 (float2*)cudaMom.even, (float2*)cudaMom.odd,
				 gridDim, blockDim,
				 (float2*)cudaMomMatrix.even, (float2*)cudaMomMatrix.odd);
		checkCudaError();



	    } //nu 
            // P5nu, Pnumu are free to be used again

	    //lepage
	    //Kernel A2
	    middle_link_kernel( (float2*)Pmu.even.data, (float2*)Pmu.odd.data,
				(float2*)Pnumu.even.data, (float2*)Pnumu.odd.data,
				(float2*)P5.even.data, (float2*)P5.odd.data,
				(float2*)Qmu.even.data, (float2*)Qmu.odd.data, // input Q matrix
				(float2*)Qnumu.even.data, (float2*)Qnumu.odd.data,
				sig, mu, Lepage,
				(float4*)cudaSiteLink.even, (float4*)cudaSiteLink.odd, cudaSiteLink, 
				(float2*)cudaMom.even, (float2*)cudaMom.odd,
				gridDim, blockDim, 
				(float2*)cudaMomMatrix.even, (float2*)cudaMomMatrix.odd); // not on a threeStaple => have to read in Qprev   
	    checkCudaError();		
	    
	    if(ThreeSt != 0)coeff = Lepage/ThreeSt ; else coeff = 0;
	    
	    side_link_kernel((float2*)P5.even.data, (float2*)P5.odd.data,
	//		     (float2*)P5nu.even.data, (float2*)P5nu.odd.data,
			     (float2*)Qmu.even.data, (float2*)Qmu.odd.data,
	//		     (float2*)Qnumu.even.data, (float2*)Qnumu.odd.data,
			     (float2*)P3.even.data, (float2*)P3.odd.data,
			     sig, mu, mLepage ,coeff,
			     (float4*)cudaSiteLink.even, (float4*)cudaSiteLink.odd, cudaSiteLink,
			     (float2*)cudaMom.even, (float2*)cudaMom.odd,
			     gridDim, blockDim,
			     (float2*)cudaMomMatrix.even, (float2*)cudaMomMatrix.odd);
	    checkCudaError();		


	    //3-link side link
	    coeff=0.;

	    side_link_kernel((float2*)P3.even.data, (float2*)P3.odd.data,
	//		     (float2*)P3mu.even.data, (float2*)P3mu.odd.data,
			     (float2*)NULL, (float2*)NULL,
	//		     (float2*)Qmu.even.data, (float2*)Qmu.odd.data,
			     (float2*)NULL, (float2*)NULL,
			     sig, mu, ThreeSt, coeff,
			     (float4*)cudaSiteLink.even, (float4*)cudaSiteLink.odd, cudaSiteLink,
			     (float2*)cudaMom.even, (float2*)cudaMom.odd,
			     gridDim, blockDim,
			     (float2*)cudaMomMatrix.even, (float2*)cudaMomMatrix.odd);
	    checkCudaError();			    




//	    //1-link and naik term	    
//	    if (!DirectLinks[mu]){
//		DirectLinks[mu]=1;
//		//kernel Z	    
//		one_and_naik_terms_kernel<<<gridDim, blockDim>>>((float2*)cudaHw.even.data, (float2*)cudaHw.odd.data,
//								 (float2*)Pmu.even.data, (float2*)Pmu.odd.data,
//								 (float2*)Pnumu.even.data, (float2*)Pnumu.odd.data,
//								 mu, OneLink.x, Naik.x, mNaik.x, 
//								 (float4*)cudaSiteLink.even, (float4*)cudaSiteLink.odd,
//								 (float2*)cudaMom.even, (float2*)cudaMom.odd);
//		checkCudaError();		
//	    }
	}//mu

    }//sig

    for(sig=0; sig<8; sig++){
      if(GOES_FORWARDS(sig)){
        compute_force_kernel( (float4*)cudaSiteLink.even, (float4*)cudaSiteLink.odd, cudaSiteLink,
                              (float2*)cudaMomMatrix.even, (float2*)cudaMomMatrix.odd,
                              sig, gridDim, blockDim,
                              (float2*)cudaMom.even, (float2*)cudaMom.odd);
      } // Only compute the force term if it goes forwards
    } // sig
    
    
}

#undef Pmu
#undef Pnumu
#undef Prhonumu
#undef P3
#undef P3mu
#undef P5
#undef P5nu
#undef P7
#undef P7rho

#undef Qmu
#undef Qnumu
#undef Qrhonumu


void
hisq_force_cuda(double eps, double weight1, double weight2, void* act_path_coeff,
		   FullOprod cudaOprod, FullGauge cudaSiteLink, FullMom cudaMom, FullGauge cudaMomMatrix, QudaGaugeParam* param)
{

    FullMatrix tempmat[4];
    for(int i=0; i<4; i++){
	tempmat[i]  = createMatQuda(param->X, param->cuda_prec);
    }

    FullMatrix tempCompmat[3];
    for(int i=0; i<3; i++){
 	tempCompmat[i] = createMatQuda(param->X, param->cuda_prec);
    }	


    if (param->cuda_prec == QUDA_DOUBLE_PRECISION){
    }else{	
	do_hisq_force_cuda( (float)eps, (float)weight1, (float)weight2, (float*)act_path_coeff,
			     cudaOprod,
			     cudaSiteLink, cudaMom, cudaMomMatrix, tempmat, tempCompmat, param);
    }
    
    for(int i=0; i<7; i++){
      freeMatQuda(tempmat[i]);
    }

    for(int i=0; i<4; i++){
      freeMatQuda(tempCompmat[i]);
    }
    return; 
}

} // namespace fermion_force
} // namespace hisq
