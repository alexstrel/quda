#include "hip/hip_runtime.h"
#include <cstdio>
#include <quda_internal.h>
#include <unitarize_quda.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <read_gauge.h>
#include <gauge_quda.h>
#include <force_common.h>


__device__ __constant__ double UNITARIZE_PI;
__device__ __constant__ double UNITARIZE_PI23;
__device__ __constant__ double UNITARIZE_EPS;

namespace hisq {

  inline __device__ __host__ int index(int i, int j) { return i*3 + j; }

  // Given a real type T, returns the corresponding complex type
  template<class T>
  struct ComplexTypeId;

  template<>
  struct ComplexTypeId<float>
  {
    typedef float2 Type;
  };

  template<>
  struct ComplexTypeId<double>
  {
    typedef double2 Type;
  };

  template<class T> 
  struct RealTypeId; 
  
  template<>
  struct RealTypeId<float2>
  {
    typedef float Type;
  };

  template<>
  struct RealTypeId<double2>
  {
    typedef double Type;
  };


 __device__ __host__
 double2 makeComplex(const double & a, const double & b){
  return make_double2(a,b);
 }
  
  __device__ __host__
  float2 makeComplex(const float & a, const float & b){
    return make_float2(a,b);
  } 
  


  template<class Cmplx>
  __device__ __host__ Cmplx operator+(const Cmplx & a, const Cmplx & b){
    return makeComplex(a.x+b.x,a.y+b.y);
  }

  template<class Cmplx>
  __device__ __host__ Cmplx operator-(const Cmplx & a, const Cmplx & b)
  {
    return makeComplex(a.x-b.x,a.y-b.y);
  }

 template<class Cmplx>
  __device__ __host__ Cmplx operator*(const Cmplx & a, const typename RealTypeId<Cmplx>::Type & scalar)
  {
    return makeComplex(a.x*scalar,a.y*scalar);
  }

  template<class Cmplx>
  __device__ __host__ Cmplx operator*(const typename RealTypeId<Cmplx>::Type & scalar, const Cmplx & b)
  {
    return operator*(b,scalar);
  }

  template<class Cmplx>
  __device__ __host__ Cmplx operator*(const Cmplx & a, const Cmplx & b)
  {
    return makeComplex(a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x);
  }

  template<class Cmplx>
  __device__ __host__ Cmplx conj(const Cmplx & a)
  {
    return makeComplex(a.x,-a.y);
  }

  template<class Cmplx>
  __device__ __host__
  const Cmplx getPreciseInverse(const Cmplx & z){
    typename RealTypeId<Cmplx>::Type ratio, max, denom;
    if( fabs(z.x) > fabs(z.y) ){ max = z.x; ratio = z.y/max; }else{ max=z.y; ratio = z.x/max; }
    denom = max*max*(1 + ratio*ratio);
    return makeComplex(z.x/denom, -z.y/denom);
  }


  // define the LinkVariable class
  template<class T> 
  class LinkVariable
  {
    public: 
	T data[9];

        // access matrix elements
        __device__ __host__ T const & operator()(int i, int j) const{
	  return data[index(i,j)];
        }

	// assign matrix elements
	__device__ __host__ T & operator()(int i, int j){
	  return data[index(i,j)];
        }
  };


  template<class T>
  __device__ __host__ const T getTrace(const LinkVariable<T> & a)
  {
    return a(0,0) + a(1,1) + a(2,2);
  }


  template<class T>
  __device__ __host__ const T getDeterminant(const LinkVariable<T> & a){
   
    T result;
    result = a(0,0)*(a(1,1)*a(2,2) - a(2,1)*a(1,2))
           - a(0,1)*(a(1,0)*a(2,2) - a(1,2)*a(2,0))
			     + a(0,2)*(a(1,0)*a(2,1) - a(1,1)*a(2,0));
	   
    return result;
  }


  template<class T>
  __device__ __host__ LinkVariable<T> operator+(const LinkVariable<T> & a, const LinkVariable<T> & b)
  {
    LinkVariable<T> result;
    for(int i=0; i<9; i++){
      result.data[i] = a.data[i] + b.data[i];
    }
     return result;
  }


  template<class T> 
  __device__ __host__ LinkVariable<T> operator-(const LinkVariable<T> & a, const LinkVariable<T> & b)
  {
    LinkVariable<T> result;
    for(int i=0; i<9; ++i){
      result.data[i] = a.data[i] - b.data[i];
    }
    return result;
  }


  template<class T, class S>
  __device__ __host__ LinkVariable<T> operator*(const S & scalar, const LinkVariable<T> & a){
    LinkVariable<T> result;
    for(int i=0; i<9; ++i){
     result.data[i] = scalar*a.data[i];
    }
    return result;
  }


  template<class T, class S>
  __device__ __host__ LinkVariable<T> operator*(const LinkVariable<T> & a, const S & scalar){
    return scalar*a;
  }


  template<class T>
  __device__ __host__
  LinkVariable<T> operator*(const LinkVariable<T> & a, const LinkVariable<T> & b)
  {
    // The compiler has a hard time unrolling nested loops,
    // so here I do it by hand. 
    // I could do something more sophisticated in the future.
    LinkVariable<T> result;
    result(0,0) = a(0,0)*b(0,0) + a(0,1)*b(1,0) + a(0,2)*b(2,0);
    result(0,1) = a(0,0)*b(0,1) + a(0,1)*b(1,1) + a(0,2)*b(2,1);
    result(0,2) = a(0,0)*b(0,2) + a(0,1)*b(1,2) + a(0,2)*b(2,2);
    result(1,0) = a(1,0)*b(0,0) + a(1,1)*b(1,0) + a(1,2)*b(2,0);
    result(1,1) = a(1,0)*b(0,1) + a(1,1)*b(1,1) + a(1,2)*b(2,1);
    result(1,2) = a(1,0)*b(0,2) + a(1,1)*b(1,2) + a(1,2)*b(2,2);
    result(2,0) = a(2,0)*b(0,0) + a(2,1)*b(1,0) + a(2,2)*b(2,0);
    result(2,1) = a(2,0)*b(0,1) + a(2,1)*b(1,1) + a(2,2)*b(2,1);
    result(2,2) = a(2,0)*b(0,2) + a(2,1)*b(1,2) + a(2,2)*b(2,2);
    return result;
  }

  template<class T>
  __device__ __host__
  LinkVariable<T> conj(const LinkVariable<T> & other){
    LinkVariable<T> result;
    for(int i=0; i<3; ++i){
      for(int j=0; j<3; ++j){
	      result(i,j) = conj(other(j,i));
      }
    }
    return result;
  }

 template<class T>
  __device__
  void loadLinkVariableFromArray(LinkVariable<T> *link, const T* const array, int dir, int idx, int stride)
  {
    for(int i=0; i<9; ++i){
      link->data[i] = array[idx + (dir*9 + i)*stride];
    }
    return;
  }

  template<class T>
  __device__
  void writeLinkVariableToArray(T* const array, const LinkVariable<T> & link,  int dir, int idx, int stride)
  {
    for(int i=0; i<9; ++i){ 
      array[idx + (dir*9 + i)*stride] = link.data[i];
    }
    return;
  }


  template<class Cmplx> 
  __device__ void reciprocalRoot(const LinkVariable<Cmplx> & q, LinkVariable<Cmplx> & res){

    LinkVariable<Cmplx> qsq, tempq;
    qsq = q*q;
    tempq = qsq*q;

    typename RealTypeId<Cmplx>::Type c[3];
    c[0] = getTrace(q).x;
    c[1] = getTrace(qsq).x/2.0;
    c[2] = getTrace(tempq).x/3.0;

    typename RealTypeId<Cmplx>::Type g[3];
    g[0] = g[1] = g[2] = c[0]/3.;
    typename RealTypeId<Cmplx>::Type r,s,theta;
    s = c[1]/3. - c[0]*c[0]/18;
    r = c[2]/2. - (c[0]/3.)*(c[1] - c[0]*c[0]/9.);

    typename RealTypeId<Cmplx>::Type cosTheta = r/sqrt(s*s*s);
    if(fabs(s) < UNITARIZE_EPS){
      cosTheta = 1.;
      s = 0.0; 
    }
    if(fabs(cosTheta)>1.0){ r>0 ? theta=0.0 : theta=UNITARIZE_PI; }
    else{ theta = acos(cosTheta); }
    theta /= 3.;
    s = 2.0*sqrt(s);
    g[0] += s*cos(theta - UNITARIZE_PI23);
    g[1] += s*cos(theta);
    g[2] += s*cos(theta + UNITARIZE_PI23);
    // At this point we have finished with the c's 
    // use these to store sqrt(g)
    c[0] = sqrt(g[0]); c[1] = sqrt(g[1]); c[2] = sqrt(g[2]);
    // done with the g's, use these to store u, v, w
    g[0] = c[0]+c[1]+c[2];
    g[1] = c[0]*c[1] + c[0]*c[2] + c[1]*c[2];
    g[2] = c[0]*c[1]*c[2];

    const typename RealTypeId<Cmplx>::Type & denominator  = g[2]*(g[0]*g[1]-g[2]); 
    c[0] = (g[0]*g[1]*g[1] - g[2]*(g[0]*g[0]+g[1]))/denominator;
    c[1] = (-g[0]*g[0]*g[0] - g[2] + 2.*g[0]*g[1])/denominator;
    c[2] =  g[0]/denominator;

    tempq = c[1]*q + c[2]*qsq;
    // Add a real scalar
    tempq(0,0).x += c[0];
    tempq(1,1).x += c[0];
    tempq(2,2).x += c[0];
	
    res = tempq;
    return;
 }
   

  // Unitarize the links using Hamilton-Cayley
  template<class Cmplx>
  __global__ void unitarize_links_hc(Cmplx* fatlink_even, Cmplx* fatlink_odd,
                                     Cmplx* ulink_even,   Cmplx* ulink_odd)
  {
    int mem_idx = blockIdx.x*blockDim.x + threadIdx.x;

    Cmplx* fatlink;
    Cmplx* ulink;

    fatlink = fatlink_even;
    ulink   = ulink_even;
    if(mem_idx >= Vh){
      mem_idx = mem_idx - Vh;
      fatlink = fatlink_odd;
      ulink   = ulink_odd;
    }

    LinkVariable<Cmplx> fat, q, rsqrt_q;
    for(int dir=0; dir<4; ++dir){

      loadLinkVariableFromArray(&fat, fatlink, dir, mem_idx, llfat_ga_stride);
      q = conj(fat)*fat;
      reciprocalRoot<Cmplx>(q, rsqrt_q);
      q = fat*rsqrt_q;

      writeLinkVariableToArray(ulink, q, dir, mem_idx, llfat_ga_stride);

    }
    return;
  } // end unitarize_links_hc






template<class Cmplx> // I need to change these to template - template parameters
__device__  __host__
void computeLinkInverse(LinkVariable<Cmplx>* uinv, const LinkVariable<Cmplx>& u)
{

   const Cmplx & det = getDeterminant(u);
   const Cmplx & det_inv = getPreciseInverse(det);

   Cmplx temp;

   temp = u(1,1)*u(2,2) - u(1,2)*u(2,1);
   (*uinv)(0,0) = (det_inv*temp);

   temp = u(0,2)*u(2,1) - u(0,1)*u(2,2);
   (*uinv)(0,1) = (temp*det_inv);

   temp = u(0,1)*u(1,2)  - u(0,2)*u(1,1);
   (*uinv)(0,2) = (temp*det_inv);

   temp = u(1,2)*u(2,0) - u(1,0)*u(2,2);
   (*uinv)(1,0) = (det_inv*temp);

   temp = u(0,0)*u(2,2) - u(0,2)*u(2,0);
   (*uinv)(1,1) = (temp*det_inv);

   temp = u(0,2)*u(1,0) - u(0,0)*u(1,2);
   (*uinv)(1,2) = (temp*det_inv);
   
   temp = u(1,0)*u(2,1) - u(1,1)*u(2,0);
   (*uinv)(2,0) = (det_inv*temp);

   temp = u(0,1)*u(2,0) - u(0,0)*u(2,1);
   (*uinv)(2,1) = (temp*det_inv);

   temp = u(0,0)*u(1,1) - u(0,1)*u(1,0);
   (*uinv)(2,2) = (temp*det_inv);
 
   return;
} 




  // simple iterative unitarization routine
  template<class Cmplx>
  __global__ void unitarize_links_si(Cmplx* fatlink_even, Cmplx* fatlink_odd,
				                             Cmplx* ulink_even,  Cmplx* ulink_odd,		
																		 int max_iters)
  {
    int mem_idx = blockIdx.x*blockDim.x + threadIdx.x;
    Cmplx* fatlink;
    Cmplx* ulink;
    fatlink = fatlink_even;
    ulink = ulink_even;
    if(mem_idx >= Vh){
      mem_idx = mem_idx - Vh;
      fatlink = fatlink_odd;
      ulink = ulink_odd;
	  }

    LinkVariable<Cmplx>  u, uinv;
 
    for(int dir=0; dir<4; ++dir){
      loadLinkVariableFromArray(&u, fatlink, dir, mem_idx, llfat_ga_stride);
      for(int i=0; i<max_iters; ++i){
        computeLinkInverse(&uinv, u);
        u = 0.5*(u + conj(uinv));	
      }
      writeLinkVariableToArray(ulink, u, dir, mem_idx, llfat_ga_stride);
    } // end loop over dirs
  } // end unitarize_links_si

} // end namespace hisq



// unitarize_init_cuda
// performs a subset of the initializations
// of llfat_init_cuda
void
unitarize_init_cuda(QudaGaugeParam* param)
{
  static int unitarize_init_cuda_flag = 0;
  if (unitarize_init_cuda_flag){
    return;
  }
  unitarize_init_cuda_flag = 1;
  
  init_kernel_cuda(param);
   
  double UNITARIZE_EPS  = 1e-5;
  hipMemcpyToSymbol(HIP_SYMBOL("UNITARIZE_EPS"), &UNITARIZE_EPS, sizeof(double));
 
  double  UNITARIZE_PI = 3.1415926535897932;
  hipMemcpyToSymbol(HIP_SYMBOL("UNITARIZE_PI"), &UNITARIZE_PI, sizeof(double));

  double UNITARIZE_PI23 = 2.*UNITARIZE_PI/3;
  hipMemcpyToSymbol(HIP_SYMBOL("UNITARIZE_PI23"), &UNITARIZE_PI23, sizeof(double));

  const int Vh = param->X[0]*param->X[1]*param->X[2]*param->X[3]/2;
  
// Need to define this so that the other routines can use it
  int site_ga_stride = param->site_ga_pad + Vh;
  hipMemcpyToSymbol(HIP_SYMBOL("site_ga_stride"), &site_ga_stride, sizeof(int));


  int llfat_ga_stride = param->llfat_ga_pad + Vh;
  hipMemcpyToSymbol(HIP_SYMBOL("llfat_ga_stride"), &llfat_ga_stride, sizeof(int));
  
  return;
}





void unitarize_cuda_hc(FullGauge cudaOutLink, FullGauge cudaInLink,
                       const QudaGaugeParam* const param)
{
  const int volume = param->X[0]*param->X[1]*param->X[2]*param->X[3];
  dim3 gridDim(volume/BLOCK_DIM,1,1);
  dim3 blockDim(BLOCK_DIM, 1, 1);
  
  
  const QudaPrecision prec = cudaInLink.precision;
  if(prec == QUDA_DOUBLE_PRECISION){
    printf("link unitarization using double precision\n");
  }else if(prec == QUDA_SINGLE_PRECISION){
    printf("link unitarization using single precision\n");
  }

  if(prec == QUDA_DOUBLE_PRECISION){
    hisq::unitarize_links_hc<<<gridDim, blockDim>>>((double2*)cudaInLink.even,  (double2*)cudaInLink.odd,
                                                    (double2*)cudaOutLink.even, (double2*)cudaOutLink.odd);

  }else{ // single precision
    hisq::unitarize_links_hc<<<gridDim, blockDim>>>((float2*)cudaInLink.even,  (float2*)cudaInLink.odd,
                                                    (float2*)cudaOutLink.even, (float2*)cudaOutLink.odd);
  }
  return;
}




void unitarize_cuda_si(FullGauge cudaOutLink, FullGauge cudaInLink,
    const QudaGaugeParam* const param, int num_its)
{
  const int volume = param->X[0]*param->X[1]*param->X[2]*param->X[3];
  dim3 gridDim(volume/BLOCK_DIM,1,1);
  dim3 blockDim(BLOCK_DIM, 1, 1);


  const QudaPrecision prec = cudaInLink.precision;
  if(prec == QUDA_DOUBLE_PRECISION){
    printf("link unitarization using double precision\n");
  }else if(prec == QUDA_SINGLE_PRECISION){
    printf("link unitarization using single precision\n");
  }

  if(prec == QUDA_DOUBLE_PRECISION){
    hisq::unitarize_links_si<<<gridDim, blockDim>>>((double2*)cudaInLink.even,  (double2*)cudaInLink.odd,
        (double2*)cudaOutLink.even, (double2*)cudaOutLink.odd,
        num_its);

  }else{ // single precision
    hisq::unitarize_links_si<<<gridDim, blockDim>>>((float2*)cudaInLink.even,  (float2*)cudaInLink.odd,
        (float2*)cudaOutLink.even, (float2*)cudaOutLink.odd,
        num_its);
  }
  return;
}
