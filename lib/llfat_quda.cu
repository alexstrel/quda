#include "hip/hip_runtime.h"
#include <stdio.h>

#include <quda_internal.h>
#include <llfat_quda.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <read_gauge.h>
#include <gauge_quda.h>
#include <force_common.h>

#if (__CUDA_ARCH__ >= 200)
#define SITE_MATRIX_LOAD_TEX 1
#define MULINK_LOAD_TEX 1
#define FATLINK_LOAD_TEX 1
#else
#define SITE_MATRIX_LOAD_TEX 0
#define MULINK_LOAD_TEX 1
#define FATLINK_LOAD_TEX 1
#endif

#define WRITE_FAT_MATRIX(gauge, dir, idx)do {			\
    gauge[idx + dir*9*llfat_ga_stride] = FAT0;			\
    gauge[idx + (dir*9+1) * llfat_ga_stride] = FAT1;			\
    gauge[idx + (dir*9+2) * llfat_ga_stride] = FAT2;			\
    gauge[idx + (dir*9+3) * llfat_ga_stride] = FAT3;			\
    gauge[idx + (dir*9+4) * llfat_ga_stride] = FAT4;		\
    gauge[idx + (dir*9+5) * llfat_ga_stride] = FAT5;		\
    gauge[idx + (dir*9+6) * llfat_ga_stride] = FAT6;		\
    gauge[idx + (dir*9+7) * llfat_ga_stride] = FAT7;		\
    gauge[idx + (dir*9+8) * llfat_ga_stride] = FAT8;} while(0)			


#define WRITE_STAPLE_MATRIX(gauge, idx)				\
  gauge[idx] = STAPLE0;						\
  gauge[idx + staple_stride] = STAPLE1;				\
  gauge[idx + 2*staple_stride] = STAPLE2;			\
  gauge[idx + 3*staple_stride] = STAPLE3;			\
  gauge[idx + 4*staple_stride] = STAPLE4;			\
  gauge[idx + 5*staple_stride] = STAPLE5;			\
  gauge[idx + 6*staple_stride] = STAPLE6;			\
  gauge[idx + 7*staple_stride] = STAPLE7;			\
  gauge[idx + 8*staple_stride] = STAPLE8;					
    

#define SCALAR_MULT_SU3_MATRIX(a, b, c) \
  c##00_re = a*b##00_re;		\
  c##00_im = a*b##00_im;		\
  c##01_re = a*b##01_re;		\
  c##01_im = a*b##01_im;		\
  c##02_re = a*b##02_re;		\
  c##02_im = a*b##02_im;		\
  c##10_re = a*b##10_re;		\
  c##10_im = a*b##10_im;		\
  c##11_re = a*b##11_re;		\
  c##11_im = a*b##11_im;		\
  c##12_re = a*b##12_re;		\
  c##12_im = a*b##12_im;		\
  c##20_re = a*b##20_re;		\
  c##20_im = a*b##20_im;		\
  c##21_re = a*b##21_re;		\
  c##21_im = a*b##21_im;		\
  c##22_re = a*b##22_re;		\
  c##22_im = a*b##22_im;		\
    

#define LOAD_MATRIX_18_SINGLE(gauge, dir, idx, var, stride)		\
  float2 var##0 = gauge[idx + dir*9*stride];				\
  float2 var##1 = gauge[idx + dir*9*stride + stride];			\
  float2 var##2 = gauge[idx + dir*9*stride + 2*stride];			\
  float2 var##3 = gauge[idx + dir*9*stride + 3*stride];			\
  float2 var##4 = gauge[idx + dir*9*stride + 4*stride];			\
  float2 var##5 = gauge[idx + dir*9*stride + 5*stride];			\
  float2 var##6 = gauge[idx + dir*9*stride + 6*stride];			\
  float2 var##7 = gauge[idx + dir*9*stride + 7*stride];			\
  float2 var##8 = gauge[idx + dir*9*stride + 8*stride];			

#define LOAD_MATRIX_18_SINGLE_TEX(gauge, dir, idx, var, stride)		\
  float2 var##0 = tex1Dfetch(gauge, idx + dir*9*stride);		\
  float2 var##1 = tex1Dfetch(gauge, idx + dir*9*stride + stride);	\
  float2 var##2 = tex1Dfetch(gauge, idx + dir*9*stride + 2*stride);	\
  float2 var##3 = tex1Dfetch(gauge, idx + dir*9*stride + 3*stride);	\
  float2 var##4 = tex1Dfetch(gauge, idx + dir*9*stride + 4*stride);	\
  float2 var##5 = tex1Dfetch(gauge, idx + dir*9*stride + 5*stride);	\
  float2 var##6 = tex1Dfetch(gauge, idx + dir*9*stride + 6*stride);	\
  float2 var##7 = tex1Dfetch(gauge, idx + dir*9*stride + 7*stride);	\
  float2 var##8 = tex1Dfetch(gauge, idx + dir*9*stride + 8*stride);	

#define LOAD_MATRIX_18_DOUBLE(gauge, dir, idx, var, stride)		\
  double2 var##0 = gauge[idx + dir*9*stride];				\
  double2 var##1 = gauge[idx + dir*9*stride + stride];			\
  double2 var##2 = gauge[idx + dir*9*stride + 2*stride];		\
  double2 var##3 = gauge[idx + dir*9*stride + 3*stride];		\
  double2 var##4 = gauge[idx + dir*9*stride + 4*stride];		\
  double2 var##5 = gauge[idx + dir*9*stride + 5*stride];		\
  double2 var##6 = gauge[idx + dir*9*stride + 6*stride];		\
  double2 var##7 = gauge[idx + dir*9*stride + 7*stride];		\
  double2 var##8 = gauge[idx + dir*9*stride + 8*stride];		

#define LOAD_MATRIX_18_DOUBLE_TEX(gauge, dir, idx, var, stride)		\
  double2 var##0 = fetch_double2(gauge, idx + dir*9*stride);		\
  double2 var##1 = fetch_double2(gauge, idx + dir*9*stride + stride);	\
  double2 var##2 = fetch_double2(gauge, idx + dir*9*stride + 2*stride);	\
  double2 var##3 = fetch_double2(gauge, idx + dir*9*stride + 3*stride);	\
  double2 var##4 = fetch_double2(gauge, idx + dir*9*stride + 4*stride);	\
  double2 var##5 = fetch_double2(gauge, idx + dir*9*stride + 5*stride);	\
  double2 var##6 = fetch_double2(gauge, idx + dir*9*stride + 6*stride);	\
  double2 var##7 = fetch_double2(gauge, idx + dir*9*stride + 7*stride);	\
  double2 var##8 = fetch_double2(gauge, idx + dir*9*stride + 8*stride);	


#define LOAD_MATRIX_12_SINGLE_DECLARE(gauge, dir, idx, var, stride)	\
  float2 var##0 = gauge[idx + dir*6*stride];				\
  float2 var##1 = gauge[idx + dir*6*stride + stride];			\
  float2 var##2 = gauge[idx + dir*6*stride + 2*stride];			\
  float2 var##3 = gauge[idx + dir*6*stride + 3*stride];			\
  float2 var##4 = gauge[idx + dir*6*stride + 4*stride];			\
  float2 var##5 = gauge[idx + dir*6*stride + 5*stride];			\
  float2 var##6, var##7, var##8;

#define LOAD_MATRIX_12_SINGLE_TEX_DECLARE(gauge, dir, idx, var, stride)	\
  float2 var##0 = tex1Dfetch(gauge, idx + dir*6*stride);		\
  float2 var##1 = tex1Dfetch(gauge, idx + dir*6*stride + stride);	\
  float2 var##2 = tex1Dfetch(gauge, idx + dir*6*stride + 2*stride);	\
  float2 var##3 = tex1Dfetch(gauge, idx + dir*6*stride + 3*stride);	\
  float2 var##4 = tex1Dfetch(gauge, idx + dir*6*stride + 4*stride);	\
  float2 var##5 = tex1Dfetch(gauge, idx + dir*6*stride + 5*stride);	\
  float2 var##6, var##7, var##8;

#define LOAD_MATRIX_18_SINGLE_DECLARE(gauge, dir, idx, var, stride)	\
  float2 var##0 = gauge[idx + dir*9*stride];				\
  float2 var##1 = gauge[idx + dir*9*stride + stride];			\
  float2 var##2 = gauge[idx + dir*9*stride + 2*stride];			\
  float2 var##3 = gauge[idx + dir*9*stride + 3*stride];			\
  float2 var##4 = gauge[idx + dir*9*stride + 4*stride];			\
  float2 var##5 = gauge[idx + dir*9*stride + 5*stride];			\
  float2 var##6 = gauge[idx + dir*9*stride + 6*stride];			\
  float2 var##7 = gauge[idx + dir*9*stride + 7*stride];			\
  float2 var##8 = gauge[idx + dir*9*stride + 8*stride];			


#define LOAD_MATRIX_18_SINGLE_TEX_DECLARE(gauge, dir, idx, var, stride)	\
  float2 var##0 = tex1Dfetch(gauge, idx + dir*9*stride);		\
  float2 var##1 = tex1Dfetch(gauge, idx + dir*9*stride + stride);	\
  float2 var##2 = tex1Dfetch(gauge, idx + dir*9*stride + 2*stride);	\
  float2 var##3 = tex1Dfetch(gauge, idx + dir*9*stride + 3*stride);	\
  float2 var##4 = tex1Dfetch(gauge, idx + dir*9*stride + 4*stride);	\
  float2 var##5 = tex1Dfetch(gauge, idx + dir*9*stride + 5*stride);	\
  float2 var##6 = tex1Dfetch(gauge, idx + dir*9*stride + 6*stride);	\
  float2 var##7 = tex1Dfetch(gauge, idx + dir*9*stride + 7*stride);	\
  float2 var##8 = tex1Dfetch(gauge, idx + dir*9*stride + 8*stride);			



#define LOAD_MATRIX_18_DOUBLE_DECLARE(gauge, dir, idx, var, stride)	\
  double2 var##0 = gauge[idx + dir*9*stride];				\
  double2 var##1 = gauge[idx + dir*9*stride + stride];			\
  double2 var##2 = gauge[idx + dir*9*stride + 2*stride];		\
  double2 var##3 = gauge[idx + dir*9*stride + 3*stride];		\
  double2 var##4 = gauge[idx + dir*9*stride + 4*stride];		\
  double2 var##5 = gauge[idx + dir*9*stride + 5*stride];		\
  double2 var##6 = gauge[idx + dir*9*stride + 6*stride];		\
  double2 var##7 = gauge[idx + dir*9*stride + 7*stride];		\
  double2 var##8 = gauge[idx + dir*9*stride + 8*stride];			


#define LOAD_MATRIX_18_DOUBLE_TEX_DECLARE(gauge, dir, idx, var, stride)	\
  double2 var##0 = fetch_double2(gauge, idx + dir*9*stride);		\
  double2 var##1 = fetch_double2(gauge, idx + dir*9*stride + stride);	\
  double2 var##2 = fetch_double2(gauge, idx + dir*9*stride + 2*stride);	\
  double2 var##3 = fetch_double2(gauge, idx + dir*9*stride + 3*stride);	\
  double2 var##4 = fetch_double2(gauge, idx + dir*9*stride + 4*stride);	\
  double2 var##5 = fetch_double2(gauge, idx + dir*9*stride + 5*stride);	\
  double2 var##6 = fetch_double2(gauge, idx + dir*9*stride + 6*stride);	\
  double2 var##7 = fetch_double2(gauge, idx + dir*9*stride + 7*stride);	\
  double2 var##8 = fetch_double2(gauge, idx + dir*9*stride + 8*stride);	


#define LOAD_MATRIX_12_DOUBLE_DECLARE(gauge, dir, idx, var, stride)		\
  double2 var##0 = gauge[idx + dir*6*stride];				\
  double2 var##1 = gauge[idx + dir*6*stride + stride];			\
  double2 var##2 = gauge[idx + dir*6*stride + 2*stride];		\
  double2 var##3 = gauge[idx + dir*6*stride + 3*stride];		\
  double2 var##4 = gauge[idx + dir*6*stride + 4*stride];		\
  double2 var##5 = gauge[idx + dir*6*stride + 5*stride];		\
  double2 var##6, var##7, var##8;


#define LOAD_MATRIX_12_DOUBLE_TEX_DECLARE(gauge, dir, idx, var, stride)	\
  double2 var##0 = fetch_double2(gauge, idx + dir*6*stride);		\
  double2 var##1 = fetch_double2(gauge, idx + dir*6*stride + stride);	\
  double2 var##2 = fetch_double2(gauge, idx + dir*6*stride + 2*stride);	\
  double2 var##3 = fetch_double2(gauge, idx + dir*6*stride + 3*stride);	\
  double2 var##4 = fetch_double2(gauge, idx + dir*6*stride + 4*stride);	\
  double2 var##5 = fetch_double2(gauge, idx + dir*6*stride + 5*stride);	\
  double2 var##6, var##7, var##8;

#define LLFAT_ADD_SU3_MATRIX(ma, mb, mc)	\
  mc##00_re = ma##00_re + mb##00_re;		\
  mc##00_im = ma##00_im + mb##00_im;		\
  mc##01_re = ma##01_re + mb##01_re;		\
  mc##01_im = ma##01_im + mb##01_im;		\
  mc##02_re = ma##02_re + mb##02_re;		\
  mc##02_im = ma##02_im + mb##02_im;		\
  mc##10_re = ma##10_re + mb##10_re;		\
  mc##10_im = ma##10_im + mb##10_im;		\
  mc##11_re = ma##11_re + mb##11_re;		\
  mc##11_im = ma##11_im + mb##11_im;		\
  mc##12_re = ma##12_re + mb##12_re;		\
  mc##12_im = ma##12_im + mb##12_im;		\
  mc##20_re = ma##20_re + mb##20_re;		\
  mc##20_im = ma##20_im + mb##20_im;		\
  mc##21_re = ma##21_re + mb##21_re;		\
  mc##21_im = ma##21_im + mb##21_im;		\
  mc##22_re = ma##22_re + mb##22_re;		\
  mc##22_im = ma##22_im + mb##22_im;		

__constant__ int dir1_array[16];
__constant__ int dir2_array[16];
__constant__ int last_proc_in_tdim;
__constant__ int first_proc_in_tdim;
__constant__ int E1, E2, E3, E4, E1h;
__constant__ int Vh_ex;
__constant__ int E2E1;
__constant__ int E3E1;
__constant__ int E3E2;
__constant__ int E3E2E1;
__constant__ int E4E2E1;
__constant__ int E4E3E1;
__constant__ int E4E3E2;

__constant__ int L1, L2, L3, L4, L1h;
__constant__ int Vh_nl;

__constant__ int L1m1, L2m1, L3m1, L4m1;
__constant__ int L2L1mL1;
__constant__ int L3L2L1mL2L1;
__constant__ int L4L3L2L1mL3L2L1;
__constant__ int L2L1;
__constant__ int L3L1;
__constant__ int L3L2;
__constant__ int L3L2L1;
__constant__ int L4L2L1;
__constant__ int L4L3L1;
__constant__ int L4L3L2;

unsigned long staple_bytes=0;

void
llfat_init_cuda(QudaGaugeParam* param)
{
  static int llfat_init_cuda_flag = 0;
  if (llfat_init_cuda_flag){
    return;
  }
  
  llfat_init_cuda_flag = 1;
  
  init_kernel_cuda(param);
  int Vh = param->X[0]*param->X[1]*param->X[2]*param->X[3]/2;
  int site_ga_stride = param->site_ga_pad + Vh;
  int staple_stride = param->staple_pad + Vh;
  int llfat_ga_stride = param->llfat_ga_pad + Vh;
  
  hipMemcpyToSymbol(HIP_SYMBOL("site_ga_stride"), &site_ga_stride, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("staple_stride"), &staple_stride, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("llfat_ga_stride"), &llfat_ga_stride, sizeof(int));
  int dir1[16];
  int dir2[16];
  for(int nu =0; nu < 4; nu++)
    for(int mu=0; mu < 4; mu++){
      if(nu == mu) continue;
      int d1, d2;
      for(d1=0; d1 < 4; d1 ++){
        if(d1 != nu && d1 != mu){
          break;
        }
      }
      dir1[nu*4+mu] = d1;

      for(d2=0; d2 < 4; d2 ++){
        if(d2 != nu && d2 != mu && d2 != d1){
          break;
        }
      }

      dir2[nu*4+mu] = d2;
    }
  
  hipMemcpyToSymbol(HIP_SYMBOL("dir1_array"), &dir1, sizeof(dir1));
  hipMemcpyToSymbol(HIP_SYMBOL("dir2_array"), &dir2, sizeof(dir2));   
  
  int first_proc_in_tdim = 0;
  int last_proc_in_tdim = 0;
  if(commCoords(3) == (commDim(3) -1)){
    last_proc_in_tdim =  1;
  }
  
  if(commCoords(3) == 0){
    first_proc_in_tdim =  1;    
  }

  hipMemcpyToSymbol(HIP_SYMBOL("last_proc_in_tdim"), &last_proc_in_tdim, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("first_proc_in_tdim"), &first_proc_in_tdim, sizeof(int));
}

void
llfat_init_cuda_ex(QudaGaugeParam* param_ex)
{
  static int llfat_init_cuda_flag = 0;
  if (llfat_init_cuda_flag){
    return;
  }
  
  llfat_init_cuda_flag = 1;
  
  init_kernel_cuda(param_ex);
  int Vh_ex = param_ex->X[0]*param_ex->X[1]*param_ex->X[2]*param_ex->X[3]/2;
  int Vh = (param_ex->X[0]-4)*(param_ex->X[1]-4)*(param_ex->X[2]-4)*(param_ex->X[3]-4)/2;
  int site_ga_stride = param_ex->site_ga_pad + Vh_ex;
  int staple_stride = param_ex->staple_pad + Vh_ex;
  int llfat_ga_stride = param_ex->llfat_ga_pad + Vh;
  
  hipMemcpyToSymbol(HIP_SYMBOL("site_ga_stride"), &site_ga_stride, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("staple_stride"), &staple_stride, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("llfat_ga_stride"), &llfat_ga_stride, sizeof(int));
  
  int first_proc_in_tdim = 0;
  int last_proc_in_tdim = 0;
  if(commCoords(3) == (commDim(3) -1)){
    last_proc_in_tdim =  1;
  }
  
  if(commCoords(3) == 0){
    first_proc_in_tdim =  1;    
  }

  hipMemcpyToSymbol(HIP_SYMBOL("last_proc_in_tdim"), &last_proc_in_tdim, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("first_proc_in_tdim"), &first_proc_in_tdim, sizeof(int));

  int E1 = param_ex->X[0];
  int E1h = E1/2;
  int E2 = param_ex->X[1];
  int E3 = param_ex->X[2];
  int E4 = param_ex->X[3];
  int E2E1 =E2*E1;
  int E3E2E1=E3*E2*E1;
  
  hipMemcpyToSymbol(HIP_SYMBOL("E1"), &E1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("E1h"), &E1h, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("E2"), &E2, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("E3"), &E3, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("E4"), &E4, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("E2E1"), &E2E1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("E3E2E1"), &E3E2E1, sizeof(int));

  hipMemcpyToSymbol(HIP_SYMBOL("Vh_ex"), &Vh_ex, sizeof(int));
  
}


void
llfat_init_cuda_nl(QudaGaugeParam* param_nl)
{
  static int llfat_init_cuda_flag = 0;
  if (llfat_init_cuda_flag){
    return;
  }
  
  llfat_init_cuda_flag = 1;
  
  init_kernel_cuda(param_nl);
  int Vh_ex = (param_nl->X[0]+2)*(param_nl->X[1]+2)*(param_nl->X[2]+2)*(param_nl->X[3]+2)/2;
  int Vh = (param_nl->X[0]-2)*(param_nl->X[1]-2)*(param_nl->X[2]-2)*(param_nl->X[3]-2)/2;
  int Vh_nl= param_nl->X[0]*param_nl->X[1]*param_nl->X[2]*param_nl->X[3]/2;
  int site_ga_stride = param_nl->site_ga_pad + Vh_nl;
  int staple_stride = param_nl->staple_pad + Vh_nl;
  int llfat_ga_stride = param_nl->llfat_ga_pad + Vh;
  
  hipMemcpyToSymbol(HIP_SYMBOL("site_ga_stride"), &site_ga_stride, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("staple_stride"), &staple_stride, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("llfat_ga_stride"), &llfat_ga_stride, sizeof(int));
  
  int first_proc_in_tdim = 0;
  int last_proc_in_tdim = 0;
  if(commCoords(3) == (commDim(3) -1)){
    last_proc_in_tdim =  1;
  }
  
  if(commCoords(3) == 0){
    first_proc_in_tdim =  1;    
  }

  hipMemcpyToSymbol(HIP_SYMBOL("last_proc_in_tdim"), &last_proc_in_tdim, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("first_proc_in_tdim"), &first_proc_in_tdim, sizeof(int));
  
  int E1 = param_nl->X[0] + 2;
  int E1h = E1/2;
  int E2 = param_nl->X[1] + 2;
  int E3 = param_nl->X[2] + 2;
  int E4 = param_nl->X[3] + 2;
  int E2E1 = E2*E1;
  int E3E1 = E3*E1;
  int E3E2 = E3*E2;
  int E3E2E1 = E3*E2*E1;
  int E4E2E1 = E4*E2*E1;
  int E4E3E1 = E4*E3*E1;
  int E4E3E2 = E4*E3*E2;    

  hipMemcpyToSymbol(HIP_SYMBOL("E1"), &E1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("E1h"), &E1h, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("E2"), &E2, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("E3"), &E3, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("E4"), &E4, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("Vh_ex"), &Vh_ex, sizeof(int));

  hipMemcpyToSymbol(HIP_SYMBOL("E2E1"), &E2E1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("E3E1"), &E3E1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("E3E2"), &E3E2, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("E3E2E1"), &E3E2E1, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("E4E2E1"), &E4E2E1, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("E4E3E1"), &E4E3E1, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("E4E3E2"), &E4E3E2, sizeof(int));    


  int L1 = param_nl->X[0];
  int L1h = L1/2;
  int L2 = param_nl->X[1];
  int L3 = param_nl->X[2];
  int L4 = param_nl->X[3];
  int L1m1 = L1-1;
  int L2m1 = L2-1;
  int L3m1 = L3-1;
  int L4m1 = L4-1;
  int L2L1mL1 = L2*L1-L1;
  int L3L2L1mL2L1 = L3*L2*L1-L2*L1;
  int L4L3L2L1mL3L2L1 = L4*L3*L2*L1 - L3*L2*L1;
  int L2L1 = L2*L1;
  int L3L1 = L3*L1;
  int L3L2 = L3*L2;
  int L3L2L1 = L3*L2*L1;
  int L4L2L1 = L4*L2*L1;
  int L4L3L1 = L4*L3*L1;
  int L4L3L2 = L4*L3*L2;  
  
  hipMemcpyToSymbol(HIP_SYMBOL("L1"), &L1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("L1h"), &L1h, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("L2"), &L2, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("L3"), &L3, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("L4"), &L4, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("Vh_nl"), &Vh_nl, sizeof(int));

  hipMemcpyToSymbol(HIP_SYMBOL("L1m1"), &L1m1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("L2m1"), &L2m1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("L3m1"), &L3m1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("L4m1"), &L4m1, sizeof(int));

  hipMemcpyToSymbol(HIP_SYMBOL("L2L1mL1"), &L2L1mL1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("L3L2L1mL2L1"), &L3L2L1mL2L1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("L4L3L2L1mL3L2L1"), &L4L3L2L1mL3L2L1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("L2L1"), &L2L1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("L3L1"), &L3L1, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("L3L2"), &L3L2, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL("L3L2L1"), &L3L2L1, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("L4L2L1"), &L4L2L1, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("L4L3L1"), &L4L3L1, sizeof(int));  
  hipMemcpyToSymbol(HIP_SYMBOL("L4L3L2"), &L4L3L2, sizeof(int));  

}



#define LLFAT_CONCAT(a,b) a##b##Kernel
#define LLFAT_CONCAT_EX(a,b) a##b##Kernel_ex
#define LLFAT_CONCAT_NL(a,b) a##b##Kernel_nl
#define LLFAT_KERNEL(a,b) LLFAT_CONCAT(a,b)
#define LLFAT_KERNEL_EX(a,b) LLFAT_CONCAT_EX(a,b)
#define LLFAT_KERNEL_NL(a,b) LLFAT_CONCAT_NL(a,b)

//precision: 0 is for double, 1 is for single

//single precision, common macro
#define PRECISION 1
#define Float  float
#define LOAD_FAT_MATRIX(gauge, dir, idx) LOAD_MATRIX_18_SINGLE(gauge, dir, idx, FAT, llfat_ga_stride)
#if (MULINK_LOAD_TEX == 1)
#define LOAD_EVEN_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE_TEX((odd_bit?muLink1TexSingle:muLink0TexSingle), dir, idx, var, staple_stride)
#define LOAD_ODD_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE_TEX((odd_bit?muLink0TexSingle:muLink1TexSingle), dir, idx, var, staple_stride)
#else
#define LOAD_EVEN_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE(mulink_even, dir, idx, var, staple_stride)
#define LOAD_ODD_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE(mulink_odd, dir, idx, var, staple_stride)
#endif

#if (FATLINK_LOAD_TEX == 1)
#define LOAD_EVEN_FAT_MATRIX(dir, idx) LOAD_MATRIX_18_SINGLE_TEX((odd_bit?fatGauge1TexSingle:fatGauge0TexSingle), dir, idx, FAT, llfat_ga_stride);
#define LOAD_ODD_FAT_MATRIX(dir, idx) LOAD_MATRIX_18_SINGLE_TEX((odd_bit?fatGauge0TexSingle:fatGauge1TexSingle), dir, idx, FAT, llfat_ga_stride);
#else
#define LOAD_EVEN_FAT_MATRIX(dir, idx) LOAD_MATRIX_18_SINGLE(fatlink_even, dir, idx, FAT, llfat_ga_stride)
#define LOAD_ODD_FAT_MATRIX(dir, idx)  LOAD_MATRIX_18_SINGLE(fatlink_odd, dir, idx, FAT, llfat_ga_stride)
#endif


//single precision, 12-reconstruct
#define SITELINK0TEX siteLink0TexSingle
#define SITELINK1TEX siteLink1TexSingle
#if (SITE_MATRIX_LOAD_TEX == 1)
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE_TEX_DECLARE((odd_bit?SITELINK1TEX:SITELINK0TEX), dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE_TEX_DECLARE((odd_bit?SITELINK0TEX:SITELINK1TEX), dir, idx, var, site_ga_stride)
#else
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE_DECLARE(sitelink_even, dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE_DECLARE(sitelink_odd, dir, idx, var, site_ga_stride)
#endif
#define LOAD_SITE_MATRIX(sitelink, dir, idx, var) LOAD_MATRIX_12_SINGLE_DECLARE(sitelink, dir, idx, var, site_ga_stride)

#define RECONSTRUCT_SITE_LINK(dir, idx, sign, var)  RECONSTRUCT_LINK_12(dir, idx, sign, var);
#define FloatN float2
#define FloatM float2
#define RECONSTRUCT 12
#include "llfat_core.h"
#undef SITELINK0TEX
#undef SITELINK1TEX
#undef LOAD_EVEN_SITE_MATRIX
#undef LOAD_ODD_SITE_MATRIX
#undef LOAD_SITE_MATRIX
#undef RECONSTRUCT_SITE_LINK
#undef FloatN
#undef FloatM
#undef RECONSTRUCT

//single precision, 18-reconstruct
#define SITELINK0TEX siteLink0TexSingle_norecon
#define SITELINK1TEX siteLink1TexSingle_norecon
#if (SITE_MATRIX_LOAD_TEX == 1)
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var)  LOAD_MATRIX_18_SINGLE_TEX_DECLARE((odd_bit?SITELINK1TEX:SITELINK0TEX), dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE_TEX_DECLARE((odd_bit?SITELINK0TEX:SITELINK1TEX), dir, idx, var, site_ga_stride)
#else
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE_DECLARE(sitelink_even, dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_SINGLE_DECLARE(sitelink_odd, dir, idx, var, site_ga_stride)
#endif
#define LOAD_SITE_MATRIX(sitelink, dir, idx, var) LOAD_MATRIX_18_SINGLE(sitelink, dir, idx, var, site_ga_stride)
#define RECONSTRUCT_SITE_LINK(dir, idx, sign, var)  
#define FloatN float2
#define FloatM float2
#define RECONSTRUCT 18
#include "llfat_core.h"
#undef SITELINK0TEX
#undef SITELINK1TEX
#undef LOAD_EVEN_SITE_MATRIX
#undef LOAD_ODD_SITE_MATRIX
#undef LOAD_SITE_MATRIX
#undef RECONSTRUCT_SITE_LINK
#undef FloatN
#undef FloatM
#undef RECONSTRUCT


#undef PRECISION
#undef Float
#undef LOAD_FAT_MATRIX
#undef LOAD_EVEN_MULINK_MATRIX
#undef LOAD_ODD_MULINK_MATRIX
#undef LOAD_EVEN_FAT_MATRIX
#undef LOAD_ODD_FAT_MATRIX


//double precision, common macro
#define PRECISION 0
#define Float double
#define LOAD_FAT_MATRIX(gauge, dir, idx) LOAD_MATRIX_18_DOUBLE(gauge, dir, idx, FAT, llfat_ga_stride)
#if (MULINK_LOAD_TEX == 1)
#define LOAD_EVEN_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE_TEX((odd_bit?muLink1TexDouble:muLink0TexDouble), dir, idx, var, staple_stride)
#define LOAD_ODD_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE_TEX((odd_bit?muLink0TexDouble:muLink1TexDouble), dir, idx, var, staple_stride)
#else
#define LOAD_EVEN_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE(mulink_even, dir, idx, var, staple_stride)
#define LOAD_ODD_MULINK_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE(mulink_odd, dir, idx, var, staple_stride)
#endif

#if (FATLINK_LOAD_TEX == 1)
#define LOAD_EVEN_FAT_MATRIX(dir, idx) LOAD_MATRIX_18_DOUBLE_TEX((odd_bit?fatGauge1TexDouble:fatGauge0TexDouble), dir, idx, FAT, llfat_ga_stride)
#define LOAD_ODD_FAT_MATRIX(dir, idx) LOAD_MATRIX_18_DOUBLE_TEX((odd_bit?fatGauge0TexDouble:fatGauge1TexDouble), dir, idx, FAT, llfat_ga_stride)
#else
#define LOAD_EVEN_FAT_MATRIX(dir, idx) LOAD_MATRIX_18_DOUBLE(fatlink_even, dir, idx, FAT, llfat_ga_stride)
#define LOAD_ODD_FAT_MATRIX(dir, idx)  LOAD_MATRIX_18_DOUBLE(fatlink_odd, dir, idx, FAT, llfat_ga_stride)
#endif

//double precision,  18-reconstruct
#define SITELINK0TEX siteLink0TexDouble
#define SITELINK1TEX siteLink1TexDouble
#if (SITE_MATRIX_LOAD_TEX == 1)
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE_TEX_DECLARE((odd_bit?SITELINK1TEX:SITELINK0TEX), dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE_TEX_DECLARE((odd_bit?SITELINK0TEX:SITELINK1TEX), dir, idx, var, site_ga_stride)
#else
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE_DECLARE(sitelink_even, dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_18_DOUBLE_DECLARE(sitelink_odd, dir, idx, var, site_ga_stride)
#endif
#define LOAD_SITE_MATRIX(sitelink, dir, idx, var) LOAD_MATRIX_18_DOUBLE(sitelink, dir, idx, var, site_ga_stride)
#define RECONSTRUCT_SITE_LINK(dir, idx, sign, var)  
#define FloatN double2
#define FloatM double2
#define RECONSTRUCT 18
#include "llfat_core.h"
#undef SITELINK0TEX
#undef SITELINK1TEX
#undef LOAD_EVEN_SITE_MATRIX
#undef LOAD_ODD_SITE_MATRIX
#undef LOAD_SITE_MATRIX
#undef RECONSTRUCT_SITE_LINK
#undef FloatN
#undef FloatM
#undef RECONSTRUCT

#if 1
//double precision, 12-reconstruct
#define SITELINK0TEX siteLink0TexDouble
#define SITELINK1TEX siteLink1TexDouble
#if (SITE_MATRIX_LOAD_TEX == 1)
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_DOUBLE_TEX_DECLARE((odd_bit?SITELINK1TEX:SITELINK0TEX), dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_DOUBLE_TEX_DECLARE((odd_bit?SITELINK0TEX:SITELINK1TEX), dir, idx, var, site_ga_stride)
#else
#define LOAD_EVEN_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_DOUBLE_DECLARE(sitelink_even, dir, idx, var, site_ga_stride)
#define LOAD_ODD_SITE_MATRIX(dir, idx, var) LOAD_MATRIX_12_DOUBLE_DECLARE(sitelink_odd, dir, idx, var, site_ga_stride)
#endif
#define LOAD_SITE_MATRIX(sitelink, dir, idx, var) LOAD_MATRIX_12_DOUBLE_DECLARE(sitelink, dir, idx, var, site_ga_stride)
#define RECONSTRUCT_SITE_LINK(dir, idx, sign, var)  RECONSTRUCT_LINK_12(dir, idx, sign, var);
#define FloatN double2
#define FloatM double2
#define RECONSTRUCT 12
#include "llfat_core.h"
#undef SITELINK0TEX
#undef SITELINK1TEX
#undef LOAD_EVEN_SITE_MATRIX
#undef LOAD_ODD_SITE_MATRIX
#undef LOAD_SITE_MATRIX
#undef RECONSTRUCT_SITE_LINK
#undef FloatN
#undef FloatM
#undef RECONSTRUCT
#endif

#undef PRECISION
#undef Float
#undef LOAD_FAT_MATRIX
#undef LOAD_EVEN_MULINK_MATRIX
#undef LOAD_ODD_MULINK_MATRIX
#undef LOAD_EVEN_FAT_MATRIX
#undef LOAD_ODD_FAT_MATRIX

#undef LLFAT_CONCAT
#undef LLFAT_CONCAT_EX
#undef LLFAT_CONCAT_NL
#undef LLFAT_KERNEL
#undef LLFAT_KERNEL_EX
#undef LLFAT_KERNEL_NL

#define UNBIND_ALL_TEXTURE do{						\
    if(prec ==QUDA_DOUBLE_PRECISION){					\
      hipUnbindTexture(siteLink0TexDouble);				\
      hipUnbindTexture(siteLink1TexDouble);				\
      hipUnbindTexture(fatGauge0TexDouble);				\
      hipUnbindTexture(fatGauge1TexDouble);				\
      hipUnbindTexture(muLink0TexDouble);				\
      hipUnbindTexture(muLink1TexDouble);				\
    }else{								\
      if(cudaSiteLink.reconstruct == QUDA_RECONSTRUCT_NO){		\
	hipUnbindTexture(siteLink0TexSingle_norecon);			\
	hipUnbindTexture(siteLink1TexSingle_norecon);			\
      }else{								\
	hipUnbindTexture(siteLink0TexSingle);				\
	hipUnbindTexture(siteLink1TexSingle);				\
      }									\
      hipUnbindTexture(fatGauge0TexSingle);				\
      hipUnbindTexture(fatGauge1TexSingle);				\
      hipUnbindTexture(muLink0TexSingle);				\
      hipUnbindTexture(muLink1TexSingle);				\
    }									\
  }while(0)

#define UNBIND_SITE_AND_FAT_LINK do{					\
    if(prec == QUDA_DOUBLE_PRECISION){					\
      hipUnbindTexture(siteLink0TexDouble);				\
      hipUnbindTexture(siteLink1TexDouble);				\
      hipUnbindTexture(fatGauge0TexDouble);				\
      hipUnbindTexture(fatGauge1TexDouble);				\
    }else {								\
      if(cudaSiteLink.reconstruct == QUDA_RECONSTRUCT_NO){		\
	hipUnbindTexture(siteLink0TexSingle_norecon);			\
	hipUnbindTexture(siteLink1TexSingle_norecon);			\
      }else{								\
	hipUnbindTexture(siteLink0TexSingle);				\
	hipUnbindTexture(siteLink1TexSingle);				\
      }									\
      hipUnbindTexture(fatGauge0TexSingle);				\
      hipUnbindTexture(fatGauge1TexSingle);				\
    }									\
  }while(0)


#define BIND_MU_LINK() do{						\
    if(prec == QUDA_DOUBLE_PRECISION){					\
      hipBindTexture(0, muLink0TexDouble, mulink_even, staple_bytes);  \
      hipBindTexture(0, muLink1TexDouble, mulink_odd, staple_bytes);	\
    }else{								\
      hipBindTexture(0, muLink0TexSingle, mulink_even, staple_bytes);  \
      hipBindTexture(0, muLink1TexSingle, mulink_odd, staple_bytes);	\
    }									\
  }while(0)

#define UNBIND_MU_LINK() do{			  \
    if(prec == QUDA_DOUBLE_PRECISION){		  \
      hipUnbindTexture(muLink0TexSingle);        \
      hipUnbindTexture(muLink1TexSingle);        \
    }else{					  \
      hipUnbindTexture(muLink0TexDouble);        \
      hipUnbindTexture(muLink1TexDouble);        \
    }						  \
  }while(0)                


#define BIND_SITE_AND_FAT_LINK do {					\
  if(prec == QUDA_DOUBLE_PRECISION){					\
    hipBindTexture(0, siteLink0TexDouble, cudaSiteLink.even, cudaSiteLink.bytes); \
    hipBindTexture(0, siteLink1TexDouble, cudaSiteLink.odd, cudaSiteLink.bytes); \
    hipBindTexture(0, fatGauge0TexDouble, cudaFatLink.even, cudaFatLink.bytes); \
    hipBindTexture(0, fatGauge1TexDouble, cudaFatLink.odd,  cudaFatLink.bytes); \
  }else{								\
    if(cudaSiteLink.reconstruct == QUDA_RECONSTRUCT_NO){		\
      hipBindTexture(0, siteLink0TexSingle_norecon, cudaSiteLink.even, cudaSiteLink.bytes); \
      hipBindTexture(0, siteLink1TexSingle_norecon, cudaSiteLink.odd, cudaSiteLink.bytes); \
    }else{								\
      hipBindTexture(0, siteLink0TexSingle, cudaSiteLink.even, cudaSiteLink.bytes); \
      hipBindTexture(0, siteLink1TexSingle, cudaSiteLink.odd, cudaSiteLink.bytes); \
    }									\
    hipBindTexture(0, fatGauge0TexSingle, cudaFatLink.even, cudaFatLink.bytes); \
    hipBindTexture(0, fatGauge1TexSingle, cudaFatLink.odd,  cudaFatLink.bytes); \
    }									\
  }while(0)

#define BIND_MU_LINK() do{						\
    if(prec == QUDA_DOUBLE_PRECISION){					\
      hipBindTexture(0, muLink0TexDouble, mulink_even, staple_bytes);	\
      hipBindTexture(0, muLink1TexDouble, mulink_odd, staple_bytes);	\
    }else{								\
      hipBindTexture(0, muLink0TexSingle, mulink_even, staple_bytes);	\
      hipBindTexture(0, muLink1TexSingle, mulink_odd, staple_bytes);	\
    }									\
  }while(0)

#define UNBIND_MU_LINK() do{						\
    if(prec == QUDA_DOUBLE_PRECISION){					\
      hipUnbindTexture(muLink0TexSingle);				\
      hipUnbindTexture(muLink1TexSingle);				\
    }else{								\
      hipUnbindTexture(muLink0TexDouble);				\
      hipUnbindTexture(muLink1TexDouble);				\
    }									\
  }while(0)								

#define BIND_SITE_AND_FAT_LINK_REVERSE do {				\
    if(prec == QUDA_DOUBLE_PRECISION){					\
      hipBindTexture(0, siteLink1TexDouble, cudaSiteLink.even, cudaSiteLink.bytes); \
      hipBindTexture(0, siteLink0TexDouble, cudaSiteLink.odd, cudaSiteLink.bytes); \
      hipBindTexture(0, fatGauge1TexDouble, cudaFatLink.even, cudaFatLink.bytes); \
      hipBindTexture(0, fatGauge0TexDouble, cudaFatLink.odd,  cudaFatLink.bytes); \
    }else{								\
      if(cudaSiteLink.reconstruct == QUDA_RECONSTRUCT_NO){		\
	hipBindTexture(0, siteLink1TexSingle_norecon, cudaSiteLink.even, cudaSiteLink.bytes); \
	hipBindTexture(0, siteLink0TexSingle_norecon, cudaSiteLink.odd, cudaSiteLink.bytes); \
      }else{								\
	hipBindTexture(0, siteLink1TexSingle, cudaSiteLink.even, cudaSiteLink.bytes); \
	hipBindTexture(0, siteLink0TexSingle, cudaSiteLink.odd, cudaSiteLink.bytes); \
      }									\
      hipBindTexture(0, fatGauge1TexSingle, cudaFatLink.even, cudaFatLink.bytes); \
      hipBindTexture(0, fatGauge0TexSingle, cudaFatLink.odd,  cudaFatLink.bytes); \
    }									\
  }while(0)



#define ENUMERATE_FUNCS(mu,nu)	switch(mu) {				\
  case 0:								\
    switch(nu){								\
    case 0:								\
      printf("ERROR: invalid direction combination\n"); exit(1);	\
      break;								\
    case 1:								\
      CALL_FUNCTION(0,1);						\
      break;								\
    case 2:								\
      CALL_FUNCTION(0,2);						\
      break;								\
    case 3:								\
      CALL_FUNCTION(0,3); 						\
      break;								\
    }									\
    break;								\
  case 1:								\
    switch(nu){								\
    case 0:								\
      CALL_FUNCTION(1,0);						\
      break;								\
    case 1:								\
      printf("ERROR: invalid direction combination\n"); exit(1);	\
      break;								\
    case 2:								\
      CALL_FUNCTION(1,2);						\
      break;								\
    case 3:								\
      CALL_FUNCTION(1,3);						\
      break;								\
    }									\
    break;								\
  case 2:								\
    switch(nu){								\
    case 0:								\
      CALL_FUNCTION(2,0);						\
      break;								\
    case 1:								\
      CALL_FUNCTION(2,1);						\
      break;								\
    case 2:								\
      printf("ERROR: invalid direction combination\n"); exit(1);	\
      break;								\
    case 3:								\
      CALL_FUNCTION(2,3);						\
      break;								\
    }									\
    break;								\
  case 3:								\
    switch(nu){								\
    case 0:								\
      CALL_FUNCTION(3,0);						\
      break;								\
    case 1:								\
      CALL_FUNCTION(3,1);						\
      break;								\
    case 2:								\
      CALL_FUNCTION(3,2);						\
      break;								\
    case 3:								\
      printf("ERROR: invalid direction combination\n"); exit(1);	\
      break;								\
    }									\
    break;								\
  }

#define ENUMERATE_FUNCS_SAVE(mu,nu, save_staple) if(save_staple){ \
    switch(mu) {							\
    case 0:								\
      switch(nu){							\
      case 0:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      case 1:								\
	CALL_FUNCTION(0,1,1); 						\
	break;								\
      case 2:								\
	CALL_FUNCTION(0,2,1);						\
	break;								\
      case 3:								\
	CALL_FUNCTION(0,3,1);						\
	break;								\
      }									\
      break;								\
    case 1:								\
      switch(nu){							\
      case 0:								\
	CALL_FUNCTION(1,0,1);						\
	break;								\
      case 1:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      case 2:								\
	CALL_FUNCTION(1,2,1);						\
	break;								\
      case 3:								\
	CALL_FUNCTION(1,3,1); 						\
	break;								\
      }									\
      break;								\
    case 2:								\
      switch(nu){							\
      case 0:								\
	CALL_FUNCTION(2,0,1);						\
	break;								\
      case 1:								\
	CALL_FUNCTION(2,1,1); 						\
	break;								\
      case 2:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      case 3:								\
	CALL_FUNCTION(2,3,1);						\
	break;								\
      }									\
      break;								\
    case 3:								\
      switch(nu){							\
      case 0:								\
	CALL_FUNCTION(3,0,1);						\
	break;								\
      case 1:								\
	CALL_FUNCTION(3,1,1);						\
	break;								\
      case 2:								\
	CALL_FUNCTION(3,2,1);						\
	break;								\
      case 3:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      }									\
      break;								\
    }									\
  }else{								\
    switch(mu) {							\
    case 0:								\
      switch(nu){							\
      case 0:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      case 1:								\
	CALL_FUNCTION(0,1,0);						\
	break;								\
      case 2:								\
	CALL_FUNCTION(0,2,0);						\
	break;								\
      case 3:								\
	CALL_FUNCTION(0,3,0);						\
	break;								\
      }									\
      break;								\
    case 1:								\
      switch(nu){							\
      case 0:								\
	CALL_FUNCTION(1,0,0);						\
	break;								\
      case 1:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      case 2:								\
	CALL_FUNCTION(1,2,0);						\
	break;								\
      case 3:								\
	CALL_FUNCTION(1,3,0); 						\
	break;								\
      }									\
      break;								\
    case 2:								\
      switch(nu){							\
      case 0:								\
	CALL_FUNCTION(2,0,0);						\
	break;								\
      case 1:								\
	CALL_FUNCTION(2,1,0);						\
	break;								\
      case 2:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      case 3:								\
	CALL_FUNCTION(2,3,0);						\
	break;								\
      }									\
      break;								\
    case 3:								\
      switch(nu){							\
      case 0:								\
	CALL_FUNCTION(3,0,0);						\
	break;								\
      case 1:								\
	CALL_FUNCTION(3,1,0);						\
	break;								\
      case 2:								\
	CALL_FUNCTION(3,2,0); 						\
	break;								\
      case 3:								\
	printf("ERROR: invalid direction combination\n"); exit(1);	\
	break;								\
      }									\
      break;								\
    }									\
  }

void siteComputeGenStapleParityKernel(void* staple_even, void* staple_odd, 
				      void* sitelink_even, void* sitelink_odd, 
				      void* fatlink_even, void* fatlink_odd,	
				      int mu, int nu, double mycoeff,
				      QudaReconstructType recon, QudaPrecision prec,
				      dim3 halfGridDim,  llfat_kernel_param_t kparam,
				      hipStream_t* stream)
{

  //compute even and odd
  
#define  CALL_FUNCTION(mu, nu)						\
  if (prec == QUDA_DOUBLE_PRECISION){					\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_siteComputeGenStapleParity18Kernel<mu,nu, 0>		\
	<<<halfGridDim, blockDim, 0, *stream>>>((double2*)staple_even, (double2*)staple_odd, \
						(double2*)sitelink_even, (double2*)sitelink_odd, \
						(double2*)fatlink_even, (double2*)fatlink_odd, \
						(double)mycoeff, kparam);	\
      do_siteComputeGenStapleParity18Kernel<mu,nu, 1>		\
	<<<halfGridDim, blockDim, 0, *stream>>>((double2*)staple_odd, (double2*)staple_even, \
						(double2*)sitelink_odd, (double2*)sitelink_even, \
						(double2*)fatlink_odd, (double2*)fatlink_even, \
						(double)mycoeff, kparam);	\
    }else{								\
      do_siteComputeGenStapleParity12Kernel<mu,nu, 0>		\
	<<<halfGridDim, blockDim, 0, *stream>>>((double2*)staple_even, (double2*)staple_odd, \
						(double2*)sitelink_even, (double2*)sitelink_odd, \
						(double2*)fatlink_even, (double2*)fatlink_odd, \
						(double)mycoeff, kparam);	\
      do_siteComputeGenStapleParity12Kernel<mu,nu, 1>		\
	<<<halfGridDim, blockDim, 0, *stream>>>((double2*)staple_odd, (double2*)staple_even, \
						(double2*)sitelink_odd, (double2*)sitelink_even, \
						(double2*)fatlink_odd, (double2*)fatlink_even, \
						(double)mycoeff, kparam);	\
    }									\
  }else {								\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_siteComputeGenStapleParity18Kernel<mu,nu, 0>		\
	<<<halfGridDim, blockDim, 0, *stream>>>((float2*)staple_even, (float2*)staple_odd, \
						(float2*)sitelink_even, (float2*)sitelink_odd, \
						(float2*)fatlink_even, (float2*)fatlink_odd, \
						(float)mycoeff, kparam);	\
      do_siteComputeGenStapleParity18Kernel<mu,nu, 1>		\
	<<<halfGridDim, blockDim, 0, *stream>>>((float2*)staple_odd, (float2*)staple_even, \
						(float2*)sitelink_odd, (float2*)sitelink_even, \
						(float2*)fatlink_odd, (float2*)fatlink_even, \
						(float)mycoeff, kparam); \
    }else{								\
      do_siteComputeGenStapleParity12Kernel<mu,nu, 0>		\
	<<<halfGridDim, blockDim, 0, *stream>>>((float2*)staple_even, (float2*)staple_odd, \
						(float2*)sitelink_even, (float2*)sitelink_odd, \
						(float2*)fatlink_even, (float2*)fatlink_odd, \
						(float)mycoeff, kparam); \
      do_siteComputeGenStapleParity12Kernel<mu,nu, 1>		\
	<<<halfGridDim, blockDim, 0, *stream>>>((float2*)staple_odd, (float2*)staple_even, \
						(float2*)sitelink_odd, (float2*)sitelink_even, \
						(float2*)fatlink_odd, (float2*)fatlink_even, \
						(float)mycoeff, kparam); \
    }									\
  }
  

  dim3 blockDim(BLOCK_DIM , 1, 1);  
  ENUMERATE_FUNCS(mu,nu);  

#undef CALL_FUNCTION
    
    
}

void siteComputeGenStapleParityKernel_ex(void* staple_even, void* staple_odd, 
					 void* sitelink_even, void* sitelink_odd, 
					 void* fatlink_even, void* fatlink_odd,	
					 int mu, int nu, double mycoeff,
					 QudaReconstructType recon, QudaPrecision prec,
					 llfat_kernel_param_t kparam)
{
  
  //compute even and odd
  dim3 halfGridDim = kparam.halfGridDim;
#define  CALL_FUNCTION(mu, nu)						\
  if (prec == QUDA_DOUBLE_PRECISION){					\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_siteComputeGenStapleParity18Kernel_ex<mu,nu, 0>		\
	<<<halfGridDim, blockDim>>>((double2*)staple_even, (double2*)staple_odd, \
				    (double2*)sitelink_even, (double2*)sitelink_odd, \
				    (double2*)fatlink_even, (double2*)fatlink_odd, \
				    (double)mycoeff, kparam);		\
      do_siteComputeGenStapleParity18Kernel_ex<mu,nu, 1>		\
	<<<halfGridDim, blockDim>>>((double2*)staple_odd, (double2*)staple_even, \
				    (double2*)sitelink_odd, (double2*)sitelink_even, \
				    (double2*)fatlink_odd, (double2*)fatlink_even, \
				    (double)mycoeff, kparam);		\
    }else{								\
      do_siteComputeGenStapleParity12Kernel_ex<mu,nu, 0>		\
	<<<halfGridDim, blockDim>>>((double2*)staple_even, (double2*)staple_odd, \
				    (double2*)sitelink_even, (double2*)sitelink_odd, \
				    (double2*)fatlink_even, (double2*)fatlink_odd, \
				    (double)mycoeff, kparam);		\
      do_siteComputeGenStapleParity12Kernel_ex<mu,nu, 1>		\
	<<<halfGridDim, blockDim>>>((double2*)staple_odd, (double2*)staple_even, \
				    (double2*)sitelink_odd, (double2*)sitelink_even, \
				    (double2*)fatlink_odd, (double2*)fatlink_even, \
				    (double)mycoeff, kparam);		\
    }									\
  }else {								\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_siteComputeGenStapleParity18Kernel_ex<mu,nu, 0>		\
	<<<halfGridDim, blockDim>>>((float2*)staple_even, (float2*)staple_odd, \
				    (float2*)sitelink_even, (float2*)sitelink_odd, \
				    (float2*)fatlink_even, (float2*)fatlink_odd, \
				    (float)mycoeff, kparam);		\
      do_siteComputeGenStapleParity18Kernel_ex<mu,nu, 1>		\
	<<<halfGridDim, blockDim>>>((float2*)staple_odd, (float2*)staple_even, \
				    (float2*)sitelink_odd, (float2*)sitelink_even, \
				    (float2*)fatlink_odd, (float2*)fatlink_even, \
				    (float)mycoeff, kparam);		\
    }else{								\
      do_siteComputeGenStapleParity12Kernel_ex<mu,nu, 0>		\
	<<<halfGridDim, blockDim>>>((float2*)staple_even, (float2*)staple_odd, \
				    (float2*)sitelink_even, (float2*)sitelink_odd, \
				    (float2*)fatlink_even, (float2*)fatlink_odd, \
				    (float)mycoeff, kparam);		\
      do_siteComputeGenStapleParity12Kernel_ex<mu,nu, 1>		\
	<<<halfGridDim, blockDim>>>((float2*)staple_odd, (float2*)staple_even, \
				    (float2*)sitelink_odd, (float2*)sitelink_even, \
				    (float2*)fatlink_odd, (float2*)fatlink_even, \
				    (float)mycoeff, kparam);		\
    }									\
  }
  
  
  dim3 blockDim(BLOCK_DIM , 1, 1);  
  ENUMERATE_FUNCS(mu,nu);  

#undef CALL_FUNCTION
    
    
}


void siteComputeGenStapleParityKernel_nl(void* staple_even, void* staple_odd, 
					 void* sitelink_even, void* sitelink_odd, 
					 void* fatlink_even, void* fatlink_odd,	
					 int mu, int nu, double mycoeff,
					 QudaReconstructType recon, QudaPrecision prec,
					 llfat_kernel_param_t kparam)
{
  
  //compute even and odd
  dim3 halfGridDim = kparam.halfGridDim;
#define  CALL_FUNCTION(mu, nu)						\
  if (prec == QUDA_DOUBLE_PRECISION){					\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_siteComputeGenStapleParity18Kernel_nl<mu,nu, 0>		\
	<<<halfGridDim, blockDim>>>((double2*)staple_even, (double2*)staple_odd, \
				    (double2*)sitelink_even, (double2*)sitelink_odd, \
				    (double2*)fatlink_even, (double2*)fatlink_odd, \
				    (double)mycoeff, kparam);		\
      do_siteComputeGenStapleParity18Kernel_nl<mu,nu, 1>		\
	<<<halfGridDim, blockDim>>>((double2*)staple_odd, (double2*)staple_even, \
				    (double2*)sitelink_odd, (double2*)sitelink_even, \
				    (double2*)fatlink_odd, (double2*)fatlink_even, \
				    (double)mycoeff, kparam);		\
    }else{								\
      do_siteComputeGenStapleParity12Kernel_nl<mu,nu, 0>		\
	<<<halfGridDim, blockDim>>>((double2*)staple_even, (double2*)staple_odd, \
				    (double2*)sitelink_even, (double2*)sitelink_odd, \
				    (double2*)fatlink_even, (double2*)fatlink_odd, \
				    (double)mycoeff, kparam);		\
      do_siteComputeGenStapleParity12Kernel_nl<mu,nu, 1>		\
	<<<halfGridDim, blockDim>>>((double2*)staple_odd, (double2*)staple_even, \
				    (double2*)sitelink_odd, (double2*)sitelink_even, \
				    (double2*)fatlink_odd, (double2*)fatlink_even, \
				    (double)mycoeff, kparam);		\
    }									\
  }else {								\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_siteComputeGenStapleParity18Kernel_nl<mu,nu, 0>		\
	<<<halfGridDim, blockDim>>>((float2*)staple_even, (float2*)staple_odd, \
				    (float2*)sitelink_even, (float2*)sitelink_odd, \
				    (float2*)fatlink_even, (float2*)fatlink_odd, \
				    (float)mycoeff, kparam);		\
      do_siteComputeGenStapleParity18Kernel_nl<mu,nu, 1>		\
	<<<halfGridDim, blockDim>>>((float2*)staple_odd, (float2*)staple_even, \
				    (float2*)sitelink_odd, (float2*)sitelink_even, \
				    (float2*)fatlink_odd, (float2*)fatlink_even, \
				    (float)mycoeff, kparam);		\
    }else{								\
      do_siteComputeGenStapleParity12Kernel_nl<mu,nu, 0>		\
	<<<halfGridDim, blockDim>>>((float2*)staple_even, (float2*)staple_odd, \
				    (float2*)sitelink_even, (float2*)sitelink_odd, \
				    (float2*)fatlink_even, (float2*)fatlink_odd, \
				    (float)mycoeff, kparam);		\
      do_siteComputeGenStapleParity12Kernel_nl<mu,nu, 1>		\
	<<<halfGridDim, blockDim>>>((float2*)staple_odd, (float2*)staple_even, \
				    (float2*)sitelink_odd, (float2*)sitelink_even, \
				    (float2*)fatlink_odd, (float2*)fatlink_even, \
				    (float)mycoeff, kparam);		\
    }									\
  }
  
  
  dim3 blockDim(BLOCK_DIM , 1, 1);  
  ENUMERATE_FUNCS(mu,nu);  

#undef CALL_FUNCTION
    
    
}


void
computeGenStapleFieldParityKernel(void* staple_even, void* staple_odd, 
				  void* sitelink_even, void* sitelink_odd,
				  void* fatlink_even, void* fatlink_odd,			    
				  void* mulink_even, void* mulink_odd, 
				  int mu, int nu, int save_staple,
				  double mycoeff,
				  QudaReconstructType recon, QudaPrecision prec,
				  dim3 halfGridDim, llfat_kernel_param_t kparam,
				  hipStream_t* stream)
{

#define  CALL_FUNCTION(mu, nu, save_staple)				\
  if (prec == QUDA_DOUBLE_PRECISION){					\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_computeGenStapleFieldParity18Kernel<mu,nu, 0, save_staple>	\
	<<<halfGridDim, blockDim, 0, *stream>>>((double2*)staple_even, (double2*)staple_odd, \
						(double2*)sitelink_even, (double2*)sitelink_odd, \
						(double2*)fatlink_even, (double2*)fatlink_odd, \
						(double2*)mulink_even, (double2*)mulink_odd, \
						(double)mycoeff, kparam); \
      do_computeGenStapleFieldParity18Kernel<mu,nu, 1, save_staple> \
	<<<halfGridDim, blockDim, 0, *stream>>>((double2*)staple_odd, (double2*)staple_even, \
						(double2*)sitelink_odd, (double2*)sitelink_even, \
						(double2*)fatlink_odd, (double2*)fatlink_even, \
						(double2*)mulink_odd, (double2*)mulink_even, \
						(double)mycoeff, kparam); \
    }else{								\
      do_computeGenStapleFieldParity12Kernel<mu,nu, 0, save_staple> \
	<<<halfGridDim, blockDim, 0, *stream>>>((double2*)staple_even, (double2*)staple_odd, \
						(double2*)sitelink_even, (double2*)sitelink_odd, \
						(double2*)fatlink_even, (double2*)fatlink_odd, \
						(double2*)mulink_even, (double2*)mulink_odd, \
						(double)mycoeff, kparam); \
      do_computeGenStapleFieldParity12Kernel<mu,nu, 1, save_staple>	\
	<<<halfGridDim, blockDim, 0, *stream>>>((double2*)staple_odd, (double2*)staple_even, \
						(double2*)sitelink_odd, (double2*)sitelink_even, \
						(double2*)fatlink_odd, (double2*)fatlink_even, \
						(double2*)mulink_odd, (double2*)mulink_even, \
						(double)mycoeff, kparam); \
    }									\
  }else{								\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_computeGenStapleFieldParity18Kernel<mu,nu, 0, save_staple>	\
	<<<halfGridDim, blockDim, 0, *stream>>>((float2*)staple_even, (float2*)staple_odd, \
						(float2*)sitelink_even, (float2*)sitelink_odd, \
						(float2*)fatlink_even, (float2*)fatlink_odd, \
						(float2*)mulink_even, (float2*)mulink_odd, \
						(float)mycoeff, kparam); \
      do_computeGenStapleFieldParity18Kernel<mu,nu, 1, save_staple>	\
	<<<halfGridDim, blockDim, 0, *stream>>>((float2*)staple_odd, (float2*)staple_even, \
						(float2*)sitelink_odd, (float2*)sitelink_even, \
						(float2*)fatlink_odd, (float2*)fatlink_even, \
						(float2*)mulink_odd, (float2*)mulink_even, \
						(float)mycoeff, kparam); \
    }else{								\
      do_computeGenStapleFieldParity12Kernel<mu,nu, 0, save_staple>	\
	<<<halfGridDim, blockDim, 0, *stream>>>((float2*)staple_even, (float2*)staple_odd, \
						(float2*)sitelink_even, (float2*)sitelink_odd, \
						(float2*)fatlink_even, (float2*)fatlink_odd, \
						(float2*)mulink_even, (float2*)mulink_odd, \
						(float)mycoeff, kparam); \
      do_computeGenStapleFieldParity12Kernel<mu,nu, 1, save_staple>	\
	<<<halfGridDim, blockDim, 0, *stream>>>((float2*)staple_odd, (float2*)staple_even, \
						(float2*)sitelink_odd, (float2*)sitelink_even, \
						(float2*)fatlink_odd, (float2*)fatlink_even, \
						(float2*)mulink_odd, (float2*)mulink_even, \
						(float)mycoeff, kparam); \
    }									\
  }
  
  BIND_MU_LINK();
  dim3 blockDim(BLOCK_DIM , 1, 1);
  ENUMERATE_FUNCS_SAVE(mu,nu,save_staple);

  UNBIND_MU_LINK();

#undef CALL_FUNCTION 
    
}


void
computeGenStapleFieldParityKernel_ex(void* staple_even, void* staple_odd, 
				     void* sitelink_even, void* sitelink_odd,
				     void* fatlink_even, void* fatlink_odd,			    
				     void* mulink_even, void* mulink_odd, 
				     int mu, int nu, int save_staple,
				     double mycoeff,
				     QudaReconstructType recon, QudaPrecision prec,
				     llfat_kernel_param_t kparam)
{

  dim3 halfGridDim= kparam.halfGridDim;
#define  CALL_FUNCTION(mu, nu, save_staple)				\
  if (prec == QUDA_DOUBLE_PRECISION){					\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_computeGenStapleFieldParity18Kernel_ex<mu,nu, 0, save_staple>	\
	<<<halfGridDim, blockDim>>>((double2*)staple_even, (double2*)staple_odd, \
				    (double2*)sitelink_even, (double2*)sitelink_odd, \
				    (double2*)fatlink_even, (double2*)fatlink_odd, \
				    (double2*)mulink_even, (double2*)mulink_odd, \
				    (double)mycoeff, kparam);			\
      do_computeGenStapleFieldParity18Kernel_ex<mu,nu, 1, save_staple>	\
	<<<halfGridDim, blockDim>>>((double2*)staple_odd, (double2*)staple_even, \
				    (double2*)sitelink_odd, (double2*)sitelink_even, \
				    (double2*)fatlink_odd, (double2*)fatlink_even, \
				    (double2*)mulink_odd, (double2*)mulink_even, \
				    (double)mycoeff, kparam);			\
    }else{								\
      do_computeGenStapleFieldParity12Kernel_ex<mu,nu, 0, save_staple>	\
	<<<halfGridDim, blockDim>>>((double2*)staple_even, (double2*)staple_odd, \
				    (double2*)sitelink_even, (double2*)sitelink_odd, \
				    (double2*)fatlink_even, (double2*)fatlink_odd, \
				    (double2*)mulink_even, (double2*)mulink_odd, \
				    (double)mycoeff, kparam);			\
      do_computeGenStapleFieldParity12Kernel_ex<mu,nu, 1, save_staple>	\
	<<<halfGridDim, blockDim>>>((double2*)staple_odd, (double2*)staple_even, \
				    (double2*)sitelink_odd, (double2*)sitelink_even, \
				    (double2*)fatlink_odd, (double2*)fatlink_even, \
				    (double2*)mulink_odd, (double2*)mulink_even, \
				    (double)mycoeff, kparam);			\
    }									\
  }else{								\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_computeGenStapleFieldParity18Kernel_ex<mu,nu, 0, save_staple>	\
	<<<halfGridDim, blockDim>>>((float2*)staple_even, (float2*)staple_odd, \
				    (float2*)sitelink_even, (float2*)sitelink_odd, \
				    (float2*)fatlink_even, (float2*)fatlink_odd, \
				    (float2*)mulink_even, (float2*)mulink_odd, \
				    (float)mycoeff, kparam);			\
      do_computeGenStapleFieldParity18Kernel_ex<mu,nu, 1, save_staple>	\
	<<<halfGridDim, blockDim>>>((float2*)staple_odd, (float2*)staple_even, \
				    (float2*)sitelink_odd, (float2*)sitelink_even, \
				    (float2*)fatlink_odd, (float2*)fatlink_even, \
				    (float2*)mulink_odd, (float2*)mulink_even, \
				    (float)mycoeff, kparam);			\
    }else{								\
      do_computeGenStapleFieldParity12Kernel_ex<mu,nu, 0, save_staple>	\
	<<<halfGridDim, blockDim>>>((float2*)staple_even, (float2*)staple_odd, \
				    (float2*)sitelink_even, (float2*)sitelink_odd, \
				    (float2*)fatlink_even, (float2*)fatlink_odd, \
				    (float2*)mulink_even, (float2*)mulink_odd, \
				    (float)mycoeff, kparam);			\
      do_computeGenStapleFieldParity12Kernel_ex<mu,nu, 1, save_staple>	\
	<<<halfGridDim, blockDim>>>((float2*)staple_odd, (float2*)staple_even, \
				    (float2*)sitelink_odd, (float2*)sitelink_even, \
				    (float2*)fatlink_odd, (float2*)fatlink_even, \
				    (float2*)mulink_odd, (float2*)mulink_even, \
				    (float)mycoeff, kparam);			\
    }									\
  }
  
  BIND_MU_LINK();
  dim3 blockDim(BLOCK_DIM , 1, 1);
  ENUMERATE_FUNCS_SAVE(mu,nu,save_staple);

  UNBIND_MU_LINK();

#undef CALL_FUNCTION 
    
}


void
computeGenStapleFieldParityKernel_nl(void* staple_even, void* staple_odd, 
				     void* sitelink_even, void* sitelink_odd,
				     void* fatlink_even, void* fatlink_odd,			    
				     void* mulink_even, void* mulink_odd, 
				     int mu, int nu, int save_staple,
				     double mycoeff,
				     QudaReconstructType recon, QudaPrecision prec,
				     llfat_kernel_param_t kparam)
{

  dim3 halfGridDim= kparam.halfGridDim;
#define  CALL_FUNCTION(mu, nu, save_staple)				\
  if (prec == QUDA_DOUBLE_PRECISION){					\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_computeGenStapleFieldParity18Kernel_nl<mu,nu, 0, save_staple>	\
	<<<halfGridDim, blockDim>>>((double2*)staple_even, (double2*)staple_odd, \
				    (double2*)sitelink_even, (double2*)sitelink_odd, \
				    (double2*)fatlink_even, (double2*)fatlink_odd, \
				    (double2*)mulink_even, (double2*)mulink_odd, \
				    (double)mycoeff, kparam);			\
      do_computeGenStapleFieldParity18Kernel_nl<mu,nu, 1, save_staple>	\
	<<<halfGridDim, blockDim>>>((double2*)staple_odd, (double2*)staple_even, \
				    (double2*)sitelink_odd, (double2*)sitelink_even, \
				    (double2*)fatlink_odd, (double2*)fatlink_even, \
				    (double2*)mulink_odd, (double2*)mulink_even, \
				    (double)mycoeff, kparam);			\
    }else{								\
      do_computeGenStapleFieldParity12Kernel_nl<mu,nu, 0, save_staple>	\
	<<<halfGridDim, blockDim>>>((double2*)staple_even, (double2*)staple_odd, \
				    (double2*)sitelink_even, (double2*)sitelink_odd, \
				    (double2*)fatlink_even, (double2*)fatlink_odd, \
				    (double2*)mulink_even, (double2*)mulink_odd, \
				    (double)mycoeff, kparam);			\
      do_computeGenStapleFieldParity12Kernel_nl<mu,nu, 1, save_staple>	\
	<<<halfGridDim, blockDim>>>((double2*)staple_odd, (double2*)staple_even, \
				    (double2*)sitelink_odd, (double2*)sitelink_even, \
				    (double2*)fatlink_odd, (double2*)fatlink_even, \
				    (double2*)mulink_odd, (double2*)mulink_even, \
				    (double)mycoeff, kparam);			\
    }									\
  }else{								\
    if(recon == QUDA_RECONSTRUCT_NO){					\
      do_computeGenStapleFieldParity18Kernel_nl<mu,nu, 0, save_staple>	\
	<<<halfGridDim, blockDim>>>((float2*)staple_even, (float2*)staple_odd, \
				    (float2*)sitelink_even, (float2*)sitelink_odd, \
				    (float2*)fatlink_even, (float2*)fatlink_odd, \
				    (float2*)mulink_even, (float2*)mulink_odd, \
				    (float)mycoeff, kparam);			\
      do_computeGenStapleFieldParity18Kernel_nl<mu,nu, 1, save_staple>	\
	<<<halfGridDim, blockDim>>>((float2*)staple_odd, (float2*)staple_even, \
				    (float2*)sitelink_odd, (float2*)sitelink_even, \
				    (float2*)fatlink_odd, (float2*)fatlink_even, \
				    (float2*)mulink_odd, (float2*)mulink_even, \
				    (float)mycoeff, kparam);			\
    }else{								\
      do_computeGenStapleFieldParity12Kernel_nl<mu,nu, 0, save_staple>	\
	<<<halfGridDim, blockDim>>>((float2*)staple_even, (float2*)staple_odd, \
				    (float2*)sitelink_even, (float2*)sitelink_odd, \
				    (float2*)fatlink_even, (float2*)fatlink_odd, \
				    (float2*)mulink_even, (float2*)mulink_odd, \
				    (float)mycoeff, kparam);			\
      do_computeGenStapleFieldParity12Kernel_nl<mu,nu, 1, save_staple>	\
	<<<halfGridDim, blockDim>>>((float2*)staple_odd, (float2*)staple_even, \
				    (float2*)sitelink_odd, (float2*)sitelink_even, \
				    (float2*)fatlink_odd, (float2*)fatlink_even, \
				    (float2*)mulink_odd, (float2*)mulink_even, \
				    (float)mycoeff, kparam);			\
    }									\
  }
  
  BIND_MU_LINK();
  dim3 blockDim(BLOCK_DIM , 1, 1);
  ENUMERATE_FUNCS_SAVE(mu,nu,save_staple);

  UNBIND_MU_LINK();

#undef CALL_FUNCTION 
    
}


void llfatOneLinkKernel(FullGauge cudaFatLink, FullGauge cudaSiteLink,
           FullStaple cudaStaple, FullStaple cudaStaple1,
           QudaGaugeParam* param, double* act_path_coeff)
{  
  QudaPrecision prec = cudaSiteLink.precision;
  QudaReconstructType recon = cudaSiteLink.reconstruct;
  
  BIND_SITE_AND_FAT_LINK;
  int volume = param->X[0]*param->X[1]*param->X[2]*param->X[3];  
  dim3 gridDim(volume/BLOCK_DIM,1,1);
  dim3 blockDim(BLOCK_DIM , 1, 1);

  staple_bytes = cudaStaple.bytes;

  if(prec == QUDA_DOUBLE_PRECISION){
    if(recon == QUDA_RECONSTRUCT_NO){
      llfatOneLink18Kernel<<<gridDim, blockDim>>>((double2*)cudaSiteLink.even, (double2*)cudaSiteLink.odd,
						  (double2*)cudaFatLink.even, (double2*)cudaFatLink.odd,
						  (double)act_path_coeff[0], (double)act_path_coeff[5]);    
    }else{
      
      llfatOneLink12Kernel<<<gridDim, blockDim>>>((double2*)cudaSiteLink.even, (double2*)cudaSiteLink.odd,
						  (double2*)cudaFatLink.even, (double2*)cudaFatLink.odd,
						  (double)act_path_coeff[0], (double)act_path_coeff[5]);    
      
    }
  }else{ //single precision
    if(recon == QUDA_RECONSTRUCT_NO){    
      llfatOneLink18Kernel<<<gridDim, blockDim>>>((float2*)cudaSiteLink.even, (float2*)cudaSiteLink.odd,
						  (float2*)cudaFatLink.even, (float2*)cudaFatLink.odd,
						  (float)act_path_coeff[0], (float)act_path_coeff[5]);    						  
    }else{
      llfatOneLink12Kernel<<<gridDim, blockDim>>>((float2*)cudaSiteLink.even, (float2*)cudaSiteLink.odd,
						  (float2*)cudaFatLink.even, (float2*)cudaFatLink.odd,
						  (float)act_path_coeff[0], (float)act_path_coeff[5]);    
    }
  }
}

void llfatOneLinkKernel_ex(FullGauge cudaFatLink, FullGauge cudaSiteLink,
			   FullStaple cudaStaple, FullStaple cudaStaple1,
			   QudaGaugeParam* param, double* act_path_coeff,
			   llfat_kernel_param_t kparam)
{  
  QudaPrecision prec = cudaSiteLink.precision;
  QudaReconstructType recon = cudaSiteLink.reconstruct;
  
  BIND_SITE_AND_FAT_LINK;
  
  dim3 gridDim;    
  dim3 blockDim(BLOCK_DIM , 1, 1);
  gridDim.x = 2* kparam.halfGridDim.x;
  gridDim.y = 1;
  gridDim.z = 1;
  staple_bytes = cudaStaple.bytes;
  
  if(prec == QUDA_DOUBLE_PRECISION){
    if(recon == QUDA_RECONSTRUCT_NO){
      llfatOneLink18Kernel_ex<<<gridDim, blockDim>>>((double2*)cudaSiteLink.even, (double2*)cudaSiteLink.odd,
						     (double2*)cudaFatLink.even, (double2*)cudaFatLink.odd,
						     (double)act_path_coeff[0], (double)act_path_coeff[5], kparam);    
    }else{
      
      llfatOneLink12Kernel_ex<<<gridDim, blockDim>>>((double2*)cudaSiteLink.even, (double2*)cudaSiteLink.odd,
						     (double2*)cudaFatLink.even, (double2*)cudaFatLink.odd,
						     (double)act_path_coeff[0], (double)act_path_coeff[5], kparam);    
      
    }
  }else{ //single precision
    if(recon == QUDA_RECONSTRUCT_NO){    
      llfatOneLink18Kernel_ex<<<gridDim, blockDim>>>((float2*)cudaSiteLink.even, (float2*)cudaSiteLink.odd,
						     (float2*)cudaFatLink.even, (float2*)cudaFatLink.odd,
						     (float)act_path_coeff[0], (float)act_path_coeff[5], kparam);    						  
    }else{
      llfatOneLink12Kernel_ex<<<gridDim, blockDim>>>((float2*)cudaSiteLink.even, (float2*)cudaSiteLink.odd,
						     (float2*)cudaFatLink.even, (float2*)cudaFatLink.odd,
						     (float)act_path_coeff[0], (float)act_path_coeff[5], kparam);    
    }
  }
}


void llfatOneLinkKernel_nl(FullGauge cudaFatLink, FullGauge cudaSiteLink,
			   FullStaple cudaStaple, FullStaple cudaStaple1,
			   QudaGaugeParam* param, double* act_path_coeff,
			   llfat_kernel_param_t kparam)
{  
  QudaPrecision prec = cudaSiteLink.precision;
  QudaReconstructType recon = cudaSiteLink.reconstruct;
  
  BIND_SITE_AND_FAT_LINK;
  
  dim3 gridDim;    
  dim3 blockDim(BLOCK_DIM , 1, 1);
  gridDim.x = 2* kparam.halfGridDim.x;
  gridDim.y = 1;
  gridDim.z = 1;
  staple_bytes = cudaStaple.bytes;
  
  if(prec == QUDA_DOUBLE_PRECISION){
    if(recon == QUDA_RECONSTRUCT_NO){
      llfatOneLink18Kernel_nl<<<gridDim, blockDim>>>((double2*)cudaSiteLink.even, (double2*)cudaSiteLink.odd,
						     (double2*)cudaFatLink.even, (double2*)cudaFatLink.odd,
						     (double)act_path_coeff[0], (double)act_path_coeff[5], kparam);    
    }else{
      
      llfatOneLink12Kernel_nl<<<gridDim, blockDim>>>((double2*)cudaSiteLink.even, (double2*)cudaSiteLink.odd,
						     (double2*)cudaFatLink.even, (double2*)cudaFatLink.odd,
						     (double)act_path_coeff[0], (double)act_path_coeff[5], kparam);    
      
    }
  }else{ //single precision
    if(recon == QUDA_RECONSTRUCT_NO){    
      llfatOneLink18Kernel_nl<<<gridDim, blockDim>>>((float2*)cudaSiteLink.even, (float2*)cudaSiteLink.odd,
						     (float2*)cudaFatLink.even, (float2*)cudaFatLink.odd,
						     (float)act_path_coeff[0], (float)act_path_coeff[5], kparam);    						  
    }else{
      llfatOneLink12Kernel_nl<<<gridDim, blockDim>>>((float2*)cudaSiteLink.even, (float2*)cudaSiteLink.odd,
						     (float2*)cudaFatLink.even, (float2*)cudaFatLink.odd,
						     (float)act_path_coeff[0], (float)act_path_coeff[5], kparam);    
    }
  }
}
